#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include "debug.h"
#include "cpu_fxn.h"
#include "device_fxn.h"
#include "global.h"

#define PI 3.141592653589793
#define running_block_size 32
#define coul2Klevin 1.6710095663e+05










__global__
void print_convergence(int n, double *x1, double *x2, double *x3, double *x4)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int i, j;
    for (j=0; j<n; j++)
    {
        for (i=index; i<n; i+=stride)
        {
            if (i==j)
            {
                // printf("index: %d\t%lf\n", i, x[i]);
                printf("%lf\t%lf\n", 1.0*x1[0]/x3[0], 1.0*x2[0]/x4[0]);
            }
        }
    }
}













double trapz(double x[], double y[], int N)
{
    int i;
    double result = 0;
    for (i=1; i<N; i++)
    {
        result = result + (x[i] - x[i-1])*(y[i] + y[i-1])/2;
    }
    return result;
}

double max(double x[], int n)
{
    int i;
    double result = x[0];
    for (i=0; i<n; i++)
    {
        if (x[i]>result)
        {
            result = x[i];
        }
    }
    return result;
}








int main(int argc, char *argv[])
{
    //To calculate the external potential field, two file strings are needed: input filename and output filename
	//define file varaiable
	FILE *fp1;
	int buffersize = 512;
	char str[buffersize];
    char conv_string[buffersize];
	//define read-in parameters
	// int Nmaxa, Nmaxb, Nmaxc;
	double La, Lb, Lc, dL;
	double alpha, beta, gamma;
    double alpha_rad, beta_rad, gamma_rad;
    int FH_signal;
    double mass, temperature[1];
    int set_running_step;
	double cutoff[1];
    int N_string[1];
    int int_N_string;
    int direction[1];
    double move_angle_degree[1], move_angle_rad[1], move_frac[1];
	int N_atom_frame[1], N_atom_adsorbate[1];
    double set_conv_trans_percent, set_conv_rot_percent;
    //define ancillary parameters
    double center_of_mass_x[1], center_of_mass_y[1], center_of_mass_z[1], total_mass_adsorbate;
    double temp_x[1], temp_y[1], temp_z[1];
    double cart_x, cart_y, cart_z;
    double cart_x_extended[1], cart_y_extended[1], cart_z_extended[1];
    int times_x[1], times_y[1], times_z[1], times;
    double a;
    // int a_N, b_N, c_N;
    double shift;
    double loc_a, loc_b, loc_c, loc_x, loc_y, loc_z, loc_u;
    double temp_frame_a, temp_frame_b, temp_frame_c;
    double temp_u;
    int i, ii, iii, iiii, j, jj, jjj, k, kk;
    double dis;
    //done!!!!!

    //read input file parameters
	fp1 = fopen(argv[1], "r");
	// fp1 = fopen("AMUWIP_charged.input", "r");
	fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
	// fscanf(fp1,"%d %d %d\n", &Nmaxa, &Nmaxb, &Nmaxc);
	fgets(str, buffersize, fp1);
	fscanf(fp1,"%lf %lf %lf %lf\n", &La, &Lb, &Lc, &dL);
	fgets(str, buffersize, fp1);
	fscanf(fp1,"%lf %lf %lf\n", &alpha, &beta, &gamma);
    alpha_rad = alpha*PI/180;
    beta_rad = beta*PI/180;
    gamma_rad = gamma*PI/180;
	fgets(str, buffersize, fp1);
    fscanf(fp1,"%lf %d %lf %lf %d\n", &cutoff[0], &FH_signal, &total_mass_adsorbate, &temperature[0], &set_running_step);
    // printf("running steps: %d\n", set_running_step);
    //read string calculation setting
    fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
    fscanf(fp1,"%d\n", &direction[0]);
    fgets(str, buffersize, fp1);
    fscanf(fp1,"%d %lf %lf\n", &N_string[0], &move_frac[0], &move_angle_degree[0]);
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%s", conv_string);
    if (strcmp(conv_string, "default")==0)
    {
        set_conv_trans_percent = 30;
        set_conv_rot_percent = 30;
    }
    fgets(str, buffersize, fp1);
    //read adsorbate information
    fgets(str, buffersize, fp1);
    // printf("%s", str);
    fgets(str, buffersize, fp1);
    // printf("%s", str);
    fscanf(fp1,"%d\n", &N_atom_adsorbate[0]);
    // printf("N_atom_adsorbate: %d\n", N_atom_adsorbate[0]);
    double x_adsorbate[N_atom_adsorbate[0]], y_adsorbate[N_atom_adsorbate[0]], z_adsorbate[N_atom_adsorbate[0]];
    double epsilon_adsorbate[N_atom_adsorbate[0]], sigma_adsorbate[N_atom_adsorbate[0]], charge_adsorbate[N_atom_adsorbate[0]], mass_adsorbate[N_atom_adsorbate[0]];
    double vector_adsorbate_x[N_atom_adsorbate[0]], vector_adsorbate_y[N_atom_adsorbate[0]], vector_adsorbate_z[N_atom_adsorbate[0]];
    fgets(str, buffersize, fp1);
    center_of_mass_x[0] = 0;
    center_of_mass_y[0] = 0;
    center_of_mass_z[0] = 0;
    for (i=0; i<N_atom_adsorbate[0]; i++)
    {
        fscanf(fp1,"%lf %lf %lf %lf %lf %lf %lf\n", &x_adsorbate[i], &y_adsorbate[i], &z_adsorbate[i], &epsilon_adsorbate[i], 
            &sigma_adsorbate[i], &charge_adsorbate[i], &mass_adsorbate[i]);
        center_of_mass_x[0] += 1.0*x_adsorbate[i]*mass_adsorbate[i]/total_mass_adsorbate;
        center_of_mass_y[0] += 1.0*y_adsorbate[i]*mass_adsorbate[i]/total_mass_adsorbate;
        center_of_mass_z[0] += 1.0*z_adsorbate[i]*mass_adsorbate[i]/total_mass_adsorbate;
    }
    // printf("center of the mass:\t%lf\t%lf\t%lf\n", center_of_mass_x, center_of_mass_y, center_of_mass_z);
    //determin the vector of each atom with respect to the center of mass
    for (i=0; i<N_atom_adsorbate[0]; i++)
    {
        vector_adsorbate_x[i] = x_adsorbate[i] - center_of_mass_x[0];
        vector_adsorbate_y[i] = y_adsorbate[i] - center_of_mass_y[0];
        vector_adsorbate_z[i] = z_adsorbate[i] - center_of_mass_z[0];
    }
    //read framework information
	fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
	fscanf(fp1,"%d\n", &N_atom_frame[0]);
	fgets(str, buffersize, fp1);

    //frac2car parameter calculation
	double frac2car_a[3];
	double frac2car_b[3];
	double frac2car_c[3];
    frac2car_a[0] = La;
    frac2car_a[1] = Lb*cos(gamma_rad);
    frac2car_a[2] = Lc*cos(beta_rad);
    frac2car_b[0] = 0;
    frac2car_b[1] = Lb*sin(gamma_rad);
    frac2car_b[2] = Lc*( (cos(alpha_rad)-cos(beta_rad)*cos(gamma_rad)) / sin(gamma_rad) );
    frac2car_c[2] = La*Lb*Lc*sqrt( 1 - pow(cos(alpha_rad),2) - pow(cos(beta_rad),2) - pow(cos(gamma_rad),2) + 2*cos(alpha_rad)*cos(beta_rad)*cos(gamma_rad) );
	frac2car_c[2] = frac2car_c[2]/(La*Lb*sin(gamma_rad));
	//done!!!!!

    //expand the cell to the size satisfied cutoff condition
    //convert the fractional cell length to cartesian value;
    frac2car(1, 0, 0, frac2car_a, frac2car_b, frac2car_c, temp_x, temp_y, temp_z);
    cart_x = temp_x[0];
    frac2car(0, 1, 0, frac2car_a, frac2car_b, frac2car_c, temp_x, temp_y, temp_z);
    cart_y = temp_y[0];
    frac2car(0, 0, 1, frac2car_a, frac2car_b, frac2car_c, temp_x, temp_y, temp_z);
    cart_z = temp_z[0];
    times_x[0] = (int) 2*cutoff[0]/cart_x + 1;
    times_y[0] = (int) 2*cutoff[0]/cart_y + 1;
    times_z[0] = (int) 2*cutoff[0]/cart_z + 1;
    times = times_x[0]*times_y[0]*times_z[0];
	double epsilon_frame[N_atom_frame[0]*times], sigma_frame[N_atom_frame[0]*times], charge_frame[N_atom_frame[0]*times], mass_frame[N_atom_frame[0]*times];
	double frac_a_frame[N_atom_frame[0]*times], frac_b_frame[N_atom_frame[0]*times], frac_c_frame[N_atom_frame[0]*times];
    for (i=0; i<N_atom_frame[0]; i++)
	{
		fscanf(fp1,"%lf %lf %lf %lf %lf %lf %lf %lf\n", &a, &sigma_frame[i], &epsilon_frame[i], &charge_frame[i], &mass_frame[i], &frac_a_frame[i], &frac_b_frame[i], &frac_c_frame[i]);
        fgets(str, buffersize, fp1);
    }
    fclose(fp1);
    pbc_expand(N_atom_frame, times_x, times_y, times_z, frac_a_frame, frac_b_frame, frac_c_frame, epsilon_frame, sigma_frame, charge_frame, mass_frame);
    frac2car(times_x[0], 0, 0, frac2car_a, frac2car_b, frac2car_c, temp_x, temp_y, temp_z);
    cart_x_extended[0] = temp_x[0];
    frac2car(0, times_y[0], 0, frac2car_a, frac2car_b, frac2car_c, temp_x, temp_y, temp_z);
    cart_y_extended[0] = temp_y[0];
    frac2car(0, 0, times_z[0], frac2car_a, frac2car_b, frac2car_c, temp_x, temp_y, temp_z);
    cart_z_extended[0] = temp_z[0];
    //done!!!!

    






    //define variables on device
    double *cart_x_extended_device, *cart_y_extended_device, *cart_z_extended_device;
    double *cutoff_device;
    int *N_atom_adsorbate_device;
    double *epsilon_adsorbate_device, *sigma_adsorbate_device, *charge_adsorbate_device;
    double *center_of_mass_x_device, *center_of_mass_y_device, *center_of_mass_z_device;
    double *vector_adsorbate_x_device, *vector_adsorbate_y_device, *vector_adsorbate_z_device;
    double *temperature_device;
    int *N_atom_frame_device;
    int *times_x_device, *times_y_device, *times_z_device;
    double *epsilon_frame_device, *sigma_frame_device, *charge_frame_device, *mass_frame_device;
    double *frac_a_frame_device, *frac_b_frame_device, *frac_c_frame_device;
    double *frac2car_a_device, *frac2car_b_device, *frac2car_c_device;
    int *direction_device;
    //allocate memory on device
    hipMalloc((void **)&cart_x_extended_device, sizeof(double));
    hipMalloc((void **)&cart_y_extended_device, sizeof(double));
    hipMalloc((void **)&cart_z_extended_device, sizeof(double));
    hipMalloc((void **)&cutoff_device, sizeof(double));
    hipMalloc((void **)&N_atom_adsorbate_device, sizeof(int));
    hipMalloc((void **)&epsilon_adsorbate_device, sizeof(double)*N_atom_adsorbate[0]);
    hipMalloc((void **)&sigma_adsorbate_device, sizeof(double)*N_atom_adsorbate[0]);
    hipMalloc((void **)&charge_adsorbate_device, sizeof(double)*N_atom_adsorbate[0]);
    hipMalloc((void **)&center_of_mass_x_device, sizeof(double));
    hipMalloc((void **)&center_of_mass_y_device, sizeof(double));
    hipMalloc((void **)&center_of_mass_z_device, sizeof(double));
    hipMalloc((void **)&vector_adsorbate_x_device, sizeof(double)*N_atom_adsorbate[0]);
    hipMalloc((void **)&vector_adsorbate_y_device, sizeof(double)*N_atom_adsorbate[0]);
    hipMalloc((void **)&vector_adsorbate_z_device, sizeof(double)*N_atom_adsorbate[0]);
    hipMalloc((void **)&temperature_device, sizeof(double));
    hipMalloc((void **)&N_atom_frame_device, sizeof(int));
    hipMalloc((void **)&times_x_device, sizeof(int));
    hipMalloc((void **)&times_y_device, sizeof(int));
    hipMalloc((void **)&times_z_device, sizeof(int));
    hipMalloc((void **)&epsilon_frame_device, sizeof(double)*N_atom_frame[0]*times);
    hipMalloc((void **)&sigma_frame_device, sizeof(double)*N_atom_frame[0]*times);
    hipMalloc((void **)&charge_frame_device, sizeof(double)*N_atom_frame[0]*times);
    hipMalloc((void **)&mass_frame_device, sizeof(double)*N_atom_frame[0]*times);
    hipMalloc((void **)&frac_a_frame_device, sizeof(double)*N_atom_frame[0]*times);
    hipMalloc((void **)&frac_b_frame_device, sizeof(double)*N_atom_frame[0]*times);
    hipMalloc((void **)&frac_c_frame_device, sizeof(double)*N_atom_frame[0]*times);
    hipMalloc((void **)&frac2car_a_device, sizeof(double)*3);
    hipMalloc((void **)&frac2car_b_device, sizeof(double)*3);
    hipMalloc((void **)&frac2car_c_device, sizeof(double)*3);
    hipMalloc((void**)&direction_device, sizeof(int));





    //define variables for Ewald summation
    double damping_a[1];
    damping_a[0] = 0.2;

    double *damping_a_device;
    hipMalloc((void **)&damping_a_device, sizeof(double));



    // //copy and transfer arrary concurrently
    // hipMemcpy(cart_x_extended_device, cart_x_extended, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(cart_y_extended_device, cart_y_extended, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(cart_z_extended_device, cart_z_extended, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(cutoff_device, cutoff, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(N_atom_adsorbate_device, N_atom_adsorbate, sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(epsilon_adsorbate_device, epsilon_adsorbate, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice);
    // hipMemcpy(sigma_adsorbate_device, sigma_adsorbate, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice);
    // hipMemcpy(center_of_mass_x_device, center_of_mass_x, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(center_of_mass_y_device, center_of_mass_y, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(center_of_mass_z_device, center_of_mass_z, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(vector_adsorbate_x_device, vector_adsorbate_x, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice);
    // hipMemcpy(vector_adsorbate_y_device, vector_adsorbate_y, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice);
    // hipMemcpy(vector_adsorbate_z_device, vector_adsorbate_z, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice);
    // hipMemcpy(temperature_device, temperature, sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(N_atom_frame_device, N_atom_frame, sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(times_x_device, times_x, sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(times_y_device, times_y, sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(times_z_device, times_z, sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(epsilon_frame_device, epsilon_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice);
    // hipMemcpy(sigma_frame_device, sigma_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice);



    //copy and transfer arrary asynchronously
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipMemcpyAsync(cart_x_extended_device, cart_x_extended, sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(cart_y_extended_device, cart_y_extended, sizeof(double), hipMemcpyHostToDevice), stream1;
    hipMemcpyAsync(cart_z_extended_device, cart_z_extended, sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(cutoff_device, cutoff, sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(N_atom_adsorbate_device, N_atom_adsorbate, sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(epsilon_adsorbate_device, epsilon_adsorbate, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(sigma_adsorbate_device, sigma_adsorbate, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(charge_adsorbate_device, charge_adsorbate, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(center_of_mass_x_device, center_of_mass_x, sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(center_of_mass_y_device, center_of_mass_y, sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(center_of_mass_z_device, center_of_mass_z, sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(vector_adsorbate_x_device, vector_adsorbate_x, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(vector_adsorbate_y_device, vector_adsorbate_y, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(vector_adsorbate_z_device, vector_adsorbate_z, sizeof(double)*N_atom_adsorbate[0], hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(temperature_device, temperature, sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(N_atom_frame_device, N_atom_frame, sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(times_x_device, times_x, sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(times_y_device, times_y, sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(times_z_device, times_z, sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(epsilon_frame_device, epsilon_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(sigma_frame_device, sigma_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(charge_frame_device, charge_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(mass_frame_device, mass_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(frac_a_frame_device, frac_a_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(frac_b_frame_device, frac_b_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(frac_c_frame_device, frac_c_frame, sizeof(double)*N_atom_frame[0]*times, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(frac2car_a_device, frac2car_a, sizeof(double)*3, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(frac2car_b_device, frac2car_b, sizeof(double)*3, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(frac2car_c_device, frac2car_c, sizeof(double)*3, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(direction_device, direction, sizeof(int), hipMemcpyHostToDevice, stream1);


    hipMemcpyAsync(damping_a_device, damping_a, sizeof(double), hipMemcpyHostToDevice, stream1);
    // hipMemcpy(direction_device, direction, sizeof(int), hipMemcpyHostToDevice);





    //check whether data is properly transferred, uncomment only when it is debugging
    // hipStreamSynchronize(stream1);
    // check_double<<<1,32>>>(1, cart_x_extended_device);
    // check_double<<<1,32>>>(1, cart_y_extended_device);
    // check_double<<<1,32>>>(1, cart_z_extended_device);
    // check_double<<<1,32>>>(1, cutoff_device);
    // check_int<<<1,32>>>(1, N_atom_adsorbate_device);
    // check_double<<<1,32>>>(N_atom_adsorbate[0], epsilon_adsorbate_device);
    // check_double<<<1,32>>>(N_atom_adsorbate[0], sigma_adsorbate_device);
    // check_double<<<1,32>>>(N_atom_adsorbate[0], charge_adsorbate_device);
    // check_double<<<1,32>>>(1, center_of_mass_x_device);
    // check_double<<<1,32>>>(1, center_of_mass_y_device);
    // check_double<<<1,32>>>(1, center_of_mass_z_device);
    // check_double<<<1,32>>>(N_atom_adsorbate[0], vector_adsorbate_x_device);
    // check_double<<<1,32>>>(N_atom_adsorbate[0], vector_adsorbate_y_device);
    // check_double<<<1,32>>>(N_atom_adsorbate[0], vector_adsorbate_z_device);
    // check_double<<<1,32>>>(1, temperature_device);
    // check_int<<<1,32>>>(1, N_atom_frame_device);
    // check_int<<<1,32>>>(1, times_x_device);
    // check_int<<<1,32>>>(1, times_y_device);
    // check_int<<<1,32>>>(1, times_z_device);
    // check_double<<<1,32>>>(N_atom_frame[0]*times, epsilon_frame_device);
    // check_double<<<1,32>>>(N_atom_frame[0]*times, sigma_frame_device);
    // check_double<<<1,32>>>(N_atom_frame[0]*times, charge_frame_device);
    // check_int<<<1,32>>>(1, direction_device);
    // hipDeviceSynchronize();
    // return 0;







    clock_t t;





    double rot_alpha_angle, rot_beta_angle, rot_gamma_angle;
    double vector_adsorbate_x_rot[N_atom_adsorbate[0]], vector_adsorbate_y_rot[N_atom_adsorbate[0]], vector_adsorbate_z_rot[N_atom_adsorbate[0]];

    double delta_angle[1];
    // delta_angle[0] = 90;
    delta_angle[0] = 60;
    double delta_grid[1];
    delta_grid[0] = 0.1;
    int N_grid[1], N_angle_alpha[1], N_angle_beta[1], N_angle_gamma[1];
    double *ini_mapping_Vext;
    double double_variable;
    // int direction = 1;
    // direction[0] = 1;
    double local_a, local_b, local_c;
    double local_x, local_y, local_z;
    double local_alpha_angle, local_beta_angle, local_gamma_angle;
    N_grid[0] = (int) (floor(1.0/delta_grid[0])+1);
    N_angle_alpha[0] = (int) (floor(360/delta_angle[0]));
    N_angle_beta[0] = (int) (floor(180/delta_angle[0]));
    N_angle_gamma[0] = (int) (floor(360/delta_angle[0]));
    ini_mapping_Vext = (double *) malloc(sizeof(double_variable)*N_grid[0]*N_grid[0]*N_angle_alpha[0]*N_angle_beta[0]*N_angle_gamma[0]);

    double a_minimum, b_minimum, c_minimum, alpha_minimum_angle, beta_minimum_angle, gamma_minimum_angle;
    double V_min;
    int minimum_signal = 0;


    hipStreamSynchronize(stream1);





    int N_points = N_grid[0]*N_grid[0]*N_angle_alpha[0]*N_angle_beta[0]*N_angle_gamma[0];




    // solution 1:
    int *N_grid_device, *N_angle_alpha_device, *N_angle_beta_device, *N_angle_gamma_device;
    double *delta_grid_device, *delta_angle_device;
    int *index_a_device, *index_b_device, *index_c_device;
    int *index_alpha_device, *index_beta_device, *index_gamma_device;
    int *index_adsorbate_device, *index_frame_device;
    double *cal_a_device, *cal_b_device, *cal_c_device;
    double *rot_alpha_rad_device, *rot_beta_rad_device, *rot_gamma_rad_device;
    double *loc_x_device, *loc_y_device, *loc_z_device;
    double *vector_adsorbate_x_rot_device, *vector_adsorbate_y_rot_device, *vector_adsorbate_z_rot_device;
    double *adsorbate_cart_x_rot_device, *adsorbate_cart_y_rot_device, *adsorbate_cart_z_rot_device;
    double *modify_frame_a_device, *modify_frame_b_device, *modify_frame_c_device;
    double *minimum_distance_device;
    double *V_total_1;
    // allocate memory
    hipMalloc((void **)&N_grid_device, sizeof(int));
    hipMalloc((void **)&N_angle_alpha_device, sizeof(int));
    hipMalloc((void **)&N_angle_beta_device, sizeof(int));
    hipMalloc((void **)&N_angle_gamma_device, sizeof(int));
    hipMalloc((void **)&delta_grid_device, sizeof(double));
    hipMalloc((void **)&delta_angle_device, sizeof(double));


    hipMalloc((void **)&index_a_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_b_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_c_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_alpha_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_beta_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_gamma_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_adsorbate_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_frame_device, sizeof(int)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);




    hipMalloc((void **)&cal_a_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&cal_b_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&cal_c_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&rot_alpha_rad_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&rot_beta_rad_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&rot_gamma_rad_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&loc_x_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&loc_y_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&loc_z_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&vector_adsorbate_x_rot_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&vector_adsorbate_y_rot_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&vector_adsorbate_z_rot_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&adsorbate_cart_x_rot_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&adsorbate_cart_y_rot_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&adsorbate_cart_z_rot_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&modify_frame_a_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&modify_frame_b_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&modify_frame_c_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&minimum_distance_device, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&V_total_1, sizeof(double)*N_points*N_atom_adsorbate[0]*N_atom_frame[0]*times);





    // memory transfer
    hipMemcpy(N_grid_device, N_grid, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(N_angle_alpha_device, N_angle_alpha, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(N_angle_beta_device, N_angle_beta, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(N_angle_gamma_device, N_angle_gamma, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(delta_grid_device, delta_grid, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(delta_angle_device, delta_angle, sizeof(double), hipMemcpyHostToDevice);

    int temp_add_frame_host[1];
    temp_add_frame_host[0] = N_atom_frame[0]*times;
    int *temp_add_frame_device;
    hipMalloc((void **)&temp_add_frame_device, sizeof(int));
    hipMemcpy(temp_add_frame_device, temp_add_frame_host, sizeof(int), hipMemcpyHostToDevice);
    // check_int<<<1,32>>>(1, temp_add_frame_device);














    int num_segments = N_points;
    int *h_offset = (int *) malloc(sizeof(int)*(num_segments+1));
    h_offset[0] = 0;
    for (i=1; i<=num_segments; i++)
    {
        h_offset[i] = i*N_atom_adsorbate[0]*N_atom_frame[0]*times;
    }
    int *d_offset;
    hipMalloc((void**)&d_offset, (num_segments+1)*sizeof(int));
    hipMemcpy(d_offset, h_offset, (num_segments+1)*sizeof(int), hipMemcpyHostToDevice);
    free(h_offset);
    double *V_out_test;
    double *V_out_print;
    V_out_print = (double *) malloc(sizeof(double)*num_segments);
    hipMalloc((void**)&V_out_test, sizeof(double)*num_segments);
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;



    // temp_1 = fopen("nohup.out", "w+");


    t = clock();


    Vext_cal<<<(int)((N_points*times*N_atom_adsorbate[0]*N_atom_frame[0]-1)/running_block_size+1),running_block_size>>>

    (N_atom_adsorbate_device, epsilon_adsorbate_device, sigma_adsorbate_device, 
    charge_adsorbate_device, 
    vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device, 
    N_atom_frame_device, epsilon_frame_device, sigma_frame_device, 
    charge_frame_device, 
    frac_a_frame_device, frac_b_frame_device, frac_c_frame_device,
    times_x_device, times_y_device, times_z_device,
    cart_x_extended_device, cart_y_extended_device, cart_z_extended_device,
    frac2car_a_device, frac2car_b_device, frac2car_c_device,
    cutoff_device, damping_a_device, 

                direction_device, 


                N_grid_device, N_angle_alpha_device, N_angle_beta_device, N_angle_gamma_device,
                delta_grid_device, delta_angle_device,
                index_a_device, index_b_device, index_c_device,
                index_alpha_device, index_beta_device, index_gamma_device,
                index_adsorbate_device, index_frame_device,

                cal_a_device, cal_b_device, cal_c_device,
                rot_alpha_rad_device, rot_beta_rad_device, rot_gamma_rad_device,
                loc_x_device, loc_y_device, loc_z_device,
                vector_adsorbate_x_rot_device, vector_adsorbate_y_rot_device, vector_adsorbate_z_rot_device,
                adsorbate_cart_x_rot_device, adsorbate_cart_y_rot_device, adsorbate_cart_z_rot_device, 
                modify_frame_a_device, modify_frame_b_device, modify_frame_c_device,
                minimum_distance_device,
                V_total_1);


    hipDeviceSynchronize();



    
    // return 0;
    // check_double_custom<<<1,32>>>(times*N_atom_adsorbate[0]*N_atom_frame[0], minimum_distance_device, V_total_1);

    

    // calculate potential energy at each grid by summing over the certain range
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_total_1, V_out_test, 
        num_segments, d_offset, d_offset+1);
    hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_total_1, V_out_test, 
        num_segments, d_offset, d_offset+1);
    hipFree(d_offset);
    hipFree(d_temp_storage);
    hipDeviceSynchronize();
    t = clock() - t;
    // printf("%lf\t", ((double)t)/CLOCKS_PER_SEC);
    // printf("gpu time: %lf\n", ((double)t)/CLOCKS_PER_SEC);
    // hipMemcpy(V_out_print, V_out_test, (num_segments)*sizeof(double), hipMemcpyDeviceToHost);
    // printf("%d\n", N_points);


    
    // find the minimum energy value and index for the configuration on the side
    d_temp_storage = NULL;
    hipcub::KeyValuePair<int, double> *min_value_index_device;
    hipMalloc((void**)&min_value_index_device, sizeof(hipcub::KeyValuePair<int, double>));
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, V_out_test, min_value_index_device, num_segments);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, V_out_test, min_value_index_device, num_segments);
    // check_key<<<1,32>>>(1, min_value_index_device);
    hipFree(d_temp_storage);

    hipFree(V_out_test);
    hipDeviceSynchronize();



    
    // return 0;




    // copy and save the information on the minimized configuration on the side




    


    


    // set up the variable for the string
    int *N_string_device;
    double *s0_a_device, *s0_b_device, *s0_c_device;
    double *s0_alpha_device, *s0_beta_device, *s0_gamma_device;
    double *s1_a_device, *s1_b_device, *s1_c_device;
    double *s1_alpha_device, *s1_beta_device, *s1_gamma_device;
    double *s1_cart_x_device, *s1_cart_y_device, *s1_cart_z_device;
    double *s1_length_coordinate_all_device, *s1_length_orientation_all_device;
    double *s1_length_coordinate_device, *s1_length_orientation_device;
    double *s1_l_abs_coordinate_device, *s1_l_abs_orientation_device;
    double *s1_length_coordinate_remap_device, *s1_length_orientation_remap_device;
    double *s1_length_coordinate_cumulation_device, *s1_length_orientation_cumulation_device;


    double *s2_a_device, *s2_b_device, *s2_c_device;
    double *s2_alpha_device, *s2_beta_device, *s2_gamma_device;
    // double *s2_a_device, *s2_b_device, *s2_c_device;
    double *s2_alpha_smooth_device, *s2_beta_smooth_device, *s2_gamma_smooth_device;


    double *s1_length_device, *s1_length_all_device, *s1_l_abs_device;
    double *s1_legnth_remap_device, *s1_length_cumulation_device;

    int  *index_s0_cal_Vext_s0_device;
    // double *index_a_cal_Vext_s0_device, *index_b_cal_Vext_s0_device, *index_c_cal_Vext_s0_device;
    // double *index_alpha_cal_Vext_s0_device, *index_beta_cal_Vext_s0_device, *index_gamma_cal_Vext_s0_device;
    int *index_adsorbate_cal_Vext_s0_device, *index_frame_cal_Vext_s0_device;

    double *a_cal_Vext_s0_device, *b_cal_Vext_s0_device, *c_cal_Vext_s0_device;
    double *alpha_rad_cal_Vext_s0_device, *beta_rad_cal_Vext_s0_device, *gamma_rad_cal_Vext_s0_device;
    double *loc_x_cal_Vext_s0_device, *loc_y_cal_Vext_s0_device, *loc_z_cal_Vext_s0_device;
    double *vector_adsorbate_x_rot_cal_Vext_s0_device, *vector_adsorbate_y_rot_cal_Vext_s0_device, *vector_adsorbate_z_rot_cal_Vext_s0_device;
    double *adsorbate_cart_x_rot_cal_Vext_s0_device, *adsorbate_cart_y_rot_cal_Vext_s0_device, *adsorbate_cart_z_rot_cal_Vext_s0_device;
    double *modify_frame_a_cal_Vext_s0_device, *modify_frame_b_cal_Vext_s0_device, *modify_frame_c_cal_Vext_s0_device;
    double *minimum_distance_cal_Vext_s0_device;
    double *V_s0_temp, *V_s0;
    double *V_s2;



    // allocate memory
    hipMalloc((void**)&N_string_device, sizeof(int));
    hipMalloc((void**)&s0_a_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s0_b_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s0_c_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s0_alpha_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s0_beta_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s0_gamma_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_a_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_b_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_c_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_alpha_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_beta_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_gamma_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_cart_x_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_cart_y_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_cart_z_device, sizeof(double)*N_string[0]);

    hipMalloc((void**)&s1_length_coordinate_all_device, sizeof(double)*N_string[0]*3);
    hipMalloc((void**)&s1_length_orientation_all_device, sizeof(double)*N_string[0]*3);
    hipMalloc((void**)&s1_length_coordinate_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_length_orientation_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_l_abs_coordinate_device, sizeof(double));
    hipMalloc((void**)&s1_l_abs_orientation_device, sizeof(double));
    hipMalloc((void**)&s1_length_coordinate_remap_device, sizeof(double)*((int) (N_string[0]*(1+N_string[0])*0.5)));
    hipMalloc((void**)&s1_length_orientation_remap_device, sizeof(double)*((int) (N_string[0]*(1+N_string[0])*0.5)));
    hipMalloc((void**)&s1_length_coordinate_cumulation_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_length_orientation_cumulation_device, sizeof(double)*N_string[0]);


    hipMalloc((void**)&s2_a_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_b_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_c_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_alpha_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_beta_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_gamma_device, sizeof(double)*N_string[0]);
    // hipMalloc((void**)&s2_a_device, sizeof(double)*N_string[0]);
    // hipMalloc((void**)&s2_b_device, sizeof(double)*N_string[0]);
    // hipMalloc((void**)&s2_c_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_alpha_smooth_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_beta_smooth_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s2_gamma_smooth_device, sizeof(double)*N_string[0]);






    hipMalloc((void**)&s1_length_device, sizeof(double)*N_string[0]);
    hipMalloc((void**)&s1_length_all_device, sizeof(double)*N_string[0]*6);
    hipMalloc((void**)&s1_l_abs_device, sizeof(double));
    hipMalloc((void**)&s1_legnth_remap_device, sizeof(double)*((int) (N_string[0]*(1+N_string[0])*0.5)));
    hipMalloc((void**)&s1_length_cumulation_device, sizeof(double)*N_string[0]);




    // copy and transfer memory
    hipMemcpy(N_string_device, N_string, sizeof(int), hipMemcpyHostToDevice);

    // check_int<<<1,32>>>(1, N_string_device);
    // check_int<<<1,32>>>(1, direction_device);


    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // initialize the string
    // printf("check\n");
    hipDeviceSynchronize();
    int signal_straight_line = 1;
    int num_inidividual_ini_extra[1];
    num_inidividual_ini_extra[0] = 5;
    double limit_transition_frac[1], limit_rotation_angle[1];
    limit_transition_frac[0] = 0.15;
    limit_rotation_angle[0] = 0;

    int *i_cal_device;



    
    

    








    // printf("%d\n", argc);
    if (argc==4)
    {
        // there is also input of initital string

        // check the compatibility of the current input string
        fp1 = fopen(argv[2], "r");
        i=0;
        while (1)
        {
            if ( fgets(str, buffersize, fp1) != NULL)
            {
                i++;
            }
            else
            {
                break;
            }
        }
        fclose(fp1);
        if (i==N_string[0])
        {
            double *temp_input_load_a, *temp_input_load_b, *temp_input_load_c;
            double *temp_input_load_alpha, *temp_input_load_beta, *temp_input_load_gamma;

            temp_input_load_a = (double *) malloc(N_string[0]*sizeof(double));
            temp_input_load_b = (double *) malloc(N_string[0]*sizeof(double));
            temp_input_load_c = (double *) malloc(N_string[0]*sizeof(double));
            temp_input_load_alpha = (double *) malloc(N_string[0]*sizeof(double));
            temp_input_load_beta = (double *) malloc(N_string[0]*sizeof(double));
            temp_input_load_gamma = (double *) malloc(N_string[0]*sizeof(double));


            fp1 = fopen(argv[2], "r");
            for (ii=0; ii<i; ii++)
            {
                fscanf(fp1, "%lf\t%lf\t%lf\t%lf\t%lf\t%lf", &temp_input_load_a[ii], &temp_input_load_b[ii], &temp_input_load_c[ii], 
                    &temp_input_load_alpha[ii], &temp_input_load_beta[ii], &temp_input_load_gamma[ii]);
                fgets(str, buffersize, fp1);
                // printf("%.10e\t%.10e\t%.10e\t%.10e\t%.10e\t%.10e\n", temp_input_load_a[ii], temp_input_load_b[ii], temp_input_load_c[ii], 
                //     temp_input_load_alpha[ii], temp_input_load_beta[ii], temp_input_load_gamma[ii]);
                // printf("%.5e\t%.5e\t%.5e\t%.5e\t%.5e\t%.5e\n", temp_input_load_a[ii], temp_input_load_b[ii], temp_input_load_c[ii], 
                //     temp_input_load_alpha[ii], temp_input_load_beta[ii], temp_input_load_gamma[ii]);


                // // debug non-stop rotation
                // temp_input_load_alpha[ii] = 0;
                // temp_input_load_beta[ii] = 0;
                // temp_input_load_gamma[ii] = 0;
                
            }
            fclose(fp1);
            hipMemcpy(s0_a_device, temp_input_load_a, N_string[0]*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(s0_b_device, temp_input_load_b, N_string[0]*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(s0_c_device, temp_input_load_c, N_string[0]*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(s0_alpha_device, temp_input_load_alpha, N_string[0]*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(s0_beta_device, temp_input_load_beta, N_string[0]*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(s0_gamma_device, temp_input_load_gamma, N_string[0]*sizeof(double), hipMemcpyHostToDevice);

        }
        else
        {
            printf("Warning!!!!\n");
            printf("Incompatible input string!!!\n");
            printf("Wrong line number!!!\n");
        }


    }
    else
    {
        if (signal_straight_line == 1)
        {
            // use straight line throught connecting the minimum energy point along the material
            ini_string_1<<<(int)((N_string[0]-1)/running_block_size+1),running_block_size>>>
            (N_string_device, cal_a_device, cal_b_device, cal_c_device, rot_alpha_rad_device, rot_beta_rad_device, rot_gamma_rad_device, 
                temp_add_frame_device, N_atom_adsorbate_device, direction_device, min_value_index_device, 
                s0_a_device, s0_b_device, s0_c_device, s0_alpha_device, s0_beta_device, s0_gamma_device);
        }
        else if (signal_straight_line == 0)
        {

            int *num_inidividual_ini_extra_device;
            double *ini_minimum_string_a_device, *ini_minimum_string_b_device, *ini_minimum_string_c_device;
            double *ini_minimum_string_alpha_device, *ini_minimum_string_beta_device, *ini_minimum_string_gamma_device;

            double *limit_transition_frac_device, *limit_rotation_angle_device;

            double *temp_partition_device;


            hipMalloc((void**)&num_inidividual_ini_extra_device, sizeof(int));
            hipMalloc((void**)&ini_minimum_string_a_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_string_b_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_string_c_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_string_alpha_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_string_beta_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_string_gamma_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&limit_transition_frac_device, sizeof(double));
            hipMalloc((void**)&limit_rotation_angle_device, sizeof(double));
            hipMalloc((void**)&i_cal_device, sizeof(double));
            hipMalloc((void**)&temp_partition_device, sizeof(double)*N_string[0]*6);



            hipMemcpy(num_inidividual_ini_extra_device, num_inidividual_ini_extra, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(limit_transition_frac_device, limit_transition_frac, sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(limit_rotation_angle_device, limit_rotation_angle, sizeof(double), hipMemcpyHostToDevice);




            copy_ini_upgrade<<<(int)((2*6)/running_block_size+1),running_block_size>>>
            (ini_minimum_string_a_device, ini_minimum_string_b_device, ini_minimum_string_c_device, 
            ini_minimum_string_alpha_device, ini_minimum_string_beta_device, ini_minimum_string_gamma_device, 

            direction_device,

            cal_a_device, cal_b_device, cal_c_device, 
            rot_alpha_rad_device, rot_beta_rad_device, rot_gamma_rad_device, 

            min_value_index_device, temp_add_frame_device, N_atom_adsorbate_device,
            num_inidividual_ini_extra_device);



            delta_angle[0] = 10;
            delta_grid[0] = 0.05;

            N_grid[0] = (int) (floor(2*limit_transition_frac[0]/delta_grid[0])+1);
            N_angle_alpha[0] = (int) (floor(2*limit_rotation_angle[0]/delta_angle[0]+1));
            N_angle_beta[0] = (int) (floor(2*limit_rotation_angle[0]/delta_angle[0]+1));
            N_angle_gamma[0] = (int) (floor(2*limit_rotation_angle[0]/delta_angle[0]+1));
            // N_angle_alpha[0] = (int) (floor(limit_rotation_angle[0]/delta_angle[0]+1));
            // N_angle_beta[0] = (int) (floor(limit_rotation_angle[0]/delta_angle[0]+1));
            // N_angle_gamma[0] = (int) (floor(limit_rotation_angle[0]/delta_angle[0]+1));





            hipMemcpy(N_grid_device, N_grid, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(N_angle_alpha_device, N_angle_alpha, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(N_angle_beta_device, N_angle_beta, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(N_angle_gamma_device, N_angle_gamma, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(delta_grid_device, delta_grid, sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(delta_angle_device, delta_angle, sizeof(double), hipMemcpyHostToDevice);



            // printf("N_points: %d\n", N_points);
            N_points = N_grid[0]*N_grid[0]*N_angle_alpha[0]*N_angle_beta[0]*N_angle_gamma[0];
            // printf("N_points: %d\n", N_points);
            // printf("N_grid: %d\n", N_grid[0]);
            // printf("N_alpha: %d\n", N_angle_alpha[0]);
            // printf("N_beta: %d\n", N_angle_beta[0]);
            // printf("N_gamma: %d\n", N_angle_gamma[0]);
            int *ini_h_offset = (int *) malloc(sizeof(int)*(N_points+1));
            ini_h_offset[0] = 0;
            for (i=1; i<=N_points; i++)
            {
                ini_h_offset[i] = i*N_atom_adsorbate[0]*N_atom_frame[0]*times;
            }
            int *ini_d_offset;
            hipMalloc((void**)&ini_d_offset, (N_points+1)*sizeof(int));
            hipMemcpy(ini_d_offset, ini_h_offset, (N_points+1)*sizeof(int), hipMemcpyHostToDevice);
            free(ini_h_offset);
            double *V_ini_test;
            hipMalloc((void**)&V_ini_test, sizeof(double)*N_points);



            for (i=1; i<=num_inidividual_ini_extra[0]; i++)
            {
                hipMemcpy(i_cal_device, &i, sizeof(int), hipMemcpyHostToDevice);

                Vext_cal_ini<<<(int)((N_points*times*N_atom_adsorbate[0]*N_atom_frame[0]-1)/running_block_size+1),running_block_size>>>

                (N_atom_adsorbate_device, epsilon_adsorbate_device, sigma_adsorbate_device,
                charge_adsorbate_device,
                vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device,
                N_atom_frame_device, epsilon_frame_device, sigma_frame_device, 
                charge_frame_device, 
                frac_a_frame_device, frac_b_frame_device, frac_c_frame_device,
                times_x_device, times_y_device, times_z_device,
                cart_x_extended_device, cart_y_extended_device, cart_z_extended_device,
                frac2car_a_device, frac2car_b_device, frac2car_c_device,
                cutoff_device, damping_a_device, 


                            direction_device,


                            N_grid_device, N_angle_alpha_device, N_angle_beta_device, N_angle_gamma_device,
                            delta_grid_device, delta_angle_device,
                            index_a_device, index_b_device, index_c_device,
                            index_alpha_device, index_beta_device, index_gamma_device,
                            index_adsorbate_device, index_frame_device,

                            limit_transition_frac_device, limit_rotation_angle_device,
                            ini_minimum_string_a_device, ini_minimum_string_b_device, ini_minimum_string_c_device, 
                            ini_minimum_string_alpha_device, ini_minimum_string_beta_device, ini_minimum_string_gamma_device,

                            i_cal_device, num_inidividual_ini_extra_device,

                            cal_a_device, cal_b_device, cal_c_device,
                            rot_alpha_rad_device, rot_beta_rad_device, rot_gamma_rad_device,
                            loc_x_device, loc_y_device, loc_z_device,
                            vector_adsorbate_x_rot_device, vector_adsorbate_y_rot_device, vector_adsorbate_z_rot_device,
                            adsorbate_cart_x_rot_device, adsorbate_cart_y_rot_device, adsorbate_cart_z_rot_device, 
                            modify_frame_a_device, modify_frame_b_device, modify_frame_c_device,
                            minimum_distance_device,
                            V_total_1);

                hipDeviceSynchronize();




                d_temp_storage = NULL;
                hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_total_1, V_ini_test, 
                    N_points, ini_d_offset, ini_d_offset+1);
                hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
                hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_total_1, V_ini_test, 
                    N_points, ini_d_offset, ini_d_offset+1);
                hipFree(d_temp_storage);
                // check_double<<<1,32>>>(N_points, V_ini_test);
                // check_double<<<1,32>>>(1, V_ini_test);
                d_temp_storage = NULL;
                hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, V_ini_test, min_value_index_device, N_points);
                hipMalloc(&d_temp_storage, temp_storage_bytes);
                hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, V_ini_test, min_value_index_device, N_points);
                // check_key<<<1,32>>>(1, min_value_index_device);
                hipFree(d_temp_storage);



                copy_ini_middle_upgrade<<<(int)((1*6)/running_block_size+1),running_block_size>>>
                (ini_minimum_string_a_device, ini_minimum_string_b_device, ini_minimum_string_c_device, 
                ini_minimum_string_alpha_device, ini_minimum_string_beta_device, ini_minimum_string_gamma_device, 

                direction_device,
                
                cal_a_device, cal_b_device, cal_c_device, 
                rot_alpha_rad_device, rot_beta_rad_device, rot_gamma_rad_device, 

                i_cal_device, 

                min_value_index_device, temp_add_frame_device, N_atom_adsorbate_device,
                num_inidividual_ini_extra_device);
            }

            hipFree(V_ini_test);

            double *ini_minimum_string_cart_x_device, *ini_minimum_string_cart_y_device, *ini_minimum_string_cart_z_device;
            double *ini_minimum_length_all_device, *ini_minimum_length_device, *ini_minimum_l_abs_device;



            hipMalloc((void**)&ini_minimum_string_cart_x_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_string_cart_y_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_string_cart_z_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_length_all_device, sizeof(double)*(num_inidividual_ini_extra[0]+2)*3);
            hipMalloc((void**)&ini_minimum_length_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));
            hipMalloc((void**)&ini_minimum_l_abs_device, sizeof(double)*1);
            // double *ini_length_a_device, *ini_length_b_device, *ini_length_c_device;
            // sizeof(double)*(num_inidividual_ini_extra[0]+2)

            // hipMalloc((void**)&ini_length_a_device, sizeof(double)*(num_inidividual_ini_extra[0]+2));


            s1_frac2cart_ini<<<(int)(((num_inidividual_ini_extra[0]+2)*3-1)/running_block_size+1),running_block_size>>>
            (num_inidividual_ini_extra_device, 

            frac2car_a_device, frac2car_b_device, frac2car_c_device,

            ini_minimum_string_a_device, ini_minimum_string_b_device, ini_minimum_string_c_device,
            ini_minimum_string_cart_x_device, ini_minimum_string_cart_y_device, ini_minimum_string_cart_z_device);

            // check_double_custom2<<<1,32>>>((num_inidividual_ini_extra[0]+2), ini_minimum_string_a_device, ini_minimum_string_b_device, ini_minimum_string_c_device, 
            // ini_minimum_string_alpha_device, ini_minimum_string_beta_device, ini_minimum_string_gamma_device);


            // check_double_custom2<<<1,32>>>((num_inidividual_ini_extra[0]+2), ini_minimum_string_cart_x_device, ini_minimum_string_cart_y_device, ini_minimum_string_cart_z_device, 
            // ini_minimum_string_alpha_device, ini_minimum_string_beta_device, ini_minimum_string_gamma_device);


            cal_length_prep_ini<<<(int)(((num_inidividual_ini_extra[0]+2)*3-1)/running_block_size+1),running_block_size>>>

            (num_inidividual_ini_extra_device, 

            ini_minimum_string_cart_x_device, ini_minimum_string_cart_y_device, ini_minimum_string_cart_z_device,
            ini_minimum_length_all_device);

            // check_double_ini<<<1,32>>>(((num_inidividual_ini_extra[0]+2)*3), ini_minimum_length_all_device);


            int *add_ini_h_offset = (int *) malloc(sizeof(int)*((num_inidividual_ini_extra[0]+2)+1));
            add_ini_h_offset[0] = 0;
            for (i=1; i<=(num_inidividual_ini_extra[0]+2); i++)
            {
                add_ini_h_offset[i] = i*3;
            }
            int *add_ini_d_offset;
            hipMalloc((void**)&add_ini_d_offset, ((num_inidividual_ini_extra[0]+2)+1)*sizeof(int));
            hipMemcpy(add_ini_d_offset, add_ini_h_offset, ((num_inidividual_ini_extra[0]+2)+1)*sizeof(int), hipMemcpyHostToDevice);
            free(add_ini_h_offset);


            d_temp_storage = NULL;
            hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, ini_minimum_length_all_device, ini_minimum_length_device, 
                (num_inidividual_ini_extra[0]+2), add_ini_d_offset, add_ini_d_offset+1);
            hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
            hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, ini_minimum_length_all_device, ini_minimum_length_device, 
                (num_inidividual_ini_extra[0]+2), add_ini_d_offset, add_ini_d_offset+1);
            hipFree(d_temp_storage);


            // check_double<<<1,32>>>((num_inidividual_ini_extra[0]+2), ini_minimum_length_device);


            ini_length_sqrt_cal<<<(int)(((num_inidividual_ini_extra[0]+2)-1)/running_block_size+1),running_block_size>>>

            (num_inidividual_ini_extra_device, ini_minimum_length_device);


            // check_double<<<1,32>>>((num_inidividual_ini_extra[0]+2), ini_minimum_length_device);

            int *sum_ini_h_offset = (int *) malloc(sizeof(int)*((1)+1));
            sum_ini_h_offset[0] = 0;
            for (i=1; i<=(1); i++)
            {
                sum_ini_h_offset[i] = (num_inidividual_ini_extra[0]+2);
            }
            int *sum_ini_d_offset;
            hipMalloc((void**)&sum_ini_d_offset, ((1)+1)*sizeof(int));
            hipMemcpy(sum_ini_d_offset, sum_ini_h_offset, ((1)+1)*sizeof(int), hipMemcpyHostToDevice);
            free(sum_ini_h_offset);

            // check_int<<<1,32>>>(2, sum_ini_d_offset);

            d_temp_storage = NULL;
            hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, ini_minimum_length_device, ini_minimum_l_abs_device, 
                (1), sum_ini_d_offset, sum_ini_d_offset+1);
            hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
            hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, ini_minimum_length_device, ini_minimum_l_abs_device, 
                (1), sum_ini_d_offset, sum_ini_d_offset+1);
            hipFree(d_temp_storage);

            // check_double<<<1,32>>>(1, ini_minimum_l_abs_device);







            ini_2_s0<<<(int)(((N_string[0]*6)-1)/running_block_size+1),running_block_size>>>

            (num_inidividual_ini_extra_device, N_string_device, 


            ini_minimum_l_abs_device, 

            ini_minimum_string_a_device, ini_minimum_string_b_device, ini_minimum_string_c_device, 
            ini_minimum_string_alpha_device, ini_minimum_string_beta_device, ini_minimum_string_gamma_device, 

            temp_partition_device, ini_minimum_length_device, 

            s0_a_device, s0_b_device, s0_c_device, 
            s0_alpha_device, s0_beta_device, s0_gamma_device);










        }
        else
        {
            printf("fatal error!!!!!\n");
            printf("fatal error!!!!!\n");
            printf("fatal error!!!!!\n");
            printf("fatal error!!!!!\n");
            printf("fatal error!!!!!\n");
            printf("fatal error!!!!!\n");
            return 0;
        }
        
    }



    













    // free memory space used to calculate the potential energy on the side
    hipFree(N_grid_device);
    hipFree(N_angle_alpha_device);
    hipFree(N_angle_beta_device);
    hipFree(N_angle_gamma_device);
    hipFree(delta_grid_device);
    hipFree(delta_angle_device);

    hipFree(index_a_device);
    hipFree(index_b_device);
    hipFree(index_c_device);
    hipFree(index_alpha_device);
    hipFree(index_beta_device);
    hipFree(index_gamma_device);
    hipFree(index_adsorbate_device);
    hipFree(index_frame_device);
    
    hipFree(loc_x_device);
    hipFree(loc_y_device);
    hipFree(loc_z_device);
    hipFree(vector_adsorbate_x_rot_device);
    hipFree(vector_adsorbate_y_rot_device);
    hipFree(vector_adsorbate_z_rot_device);
    hipFree(adsorbate_cart_x_rot_device);
    hipFree(adsorbate_cart_y_rot_device);
    hipFree(adsorbate_cart_z_rot_device);
    hipFree(modify_frame_a_device);
    hipFree(modify_frame_b_device);
    hipFree(modify_frame_c_device);
    hipFree(minimum_distance_device);
    hipFree(V_total_1);





    
    // free memory space used to calculate the potential energy on the side
    hipFree(cal_a_device);
    hipFree(cal_b_device);
    hipFree(cal_c_device);
    hipFree(rot_alpha_rad_device);
    hipFree(rot_beta_rad_device);
    hipFree(rot_gamma_rad_device);



    // calculate energy for the potential along the string without calculating anything extra related to the derivative
    hipMalloc((void **)&index_s0_cal_Vext_s0_device, sizeof(int)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_adsorbate_cal_Vext_s0_device, sizeof(int)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&index_frame_cal_Vext_s0_device, sizeof(int)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&a_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&b_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&c_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&alpha_rad_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&beta_rad_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&gamma_rad_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&loc_x_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&loc_y_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&loc_z_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&vector_adsorbate_x_rot_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&vector_adsorbate_y_rot_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&vector_adsorbate_z_rot_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&adsorbate_cart_x_rot_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&adsorbate_cart_y_rot_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&adsorbate_cart_z_rot_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&modify_frame_a_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&modify_frame_b_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&modify_frame_c_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&minimum_distance_cal_Vext_s0_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&V_s0_temp, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times);
    hipMalloc((void **)&V_s0, sizeof(double)*N_string[0]);
    hipMalloc((void **)&V_s2, sizeof(double)*N_string[0]);





    Vext_cal_s0<<<(int)((N_string[0]*times*N_atom_adsorbate[0]*N_atom_frame[0]-1)/running_block_size+1),running_block_size>>>
    (N_atom_adsorbate_device, epsilon_adsorbate_device, sigma_adsorbate_device, 
                charge_adsorbate_device, 
    vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device,
                N_atom_frame_device, epsilon_frame_device, sigma_frame_device, 
                charge_frame_device, 
                frac_a_frame_device, frac_b_frame_device, frac_c_frame_device, 
                times_x_device, times_y_device, times_z_device, 
                cart_x_extended_device, cart_y_extended_device, cart_z_extended_device, 
                frac2car_a_device, frac2car_b_device, frac2car_c_device, 
                cutoff_device, damping_a_device, 
                temp_add_frame_device, 


                // int *N_grid_device, int *N_angle_alpha_device, int *N_angle_beta_device, int *N_angle_gamma_device,
                // double *delta_grid_device, double *delta_angle_device,
                N_string_device,

                s0_a_device, s0_b_device, s0_c_device, 
                s0_alpha_device, s0_beta_device, s0_gamma_device,


                index_s0_cal_Vext_s0_device,
                // int *index_a_cal_Vext_s0_device, int *index_b_cal_Vext_s0_device, int *index_c_cal_Vext_s0_device,
                // int *index_alpha_cal_Vext_s0_device, int *index_beta_cal_Vext_s0_device, int *index_gamma_cal_Vext_s0_device,
                index_adsorbate_cal_Vext_s0_device, index_frame_cal_Vext_s0_device,

                a_cal_Vext_s0_device, b_cal_Vext_s0_device, c_cal_Vext_s0_device,
                alpha_rad_cal_Vext_s0_device, beta_rad_cal_Vext_s0_device, gamma_rad_cal_Vext_s0_device,
                loc_x_cal_Vext_s0_device, loc_y_cal_Vext_s0_device, loc_z_cal_Vext_s0_device,
                vector_adsorbate_x_rot_cal_Vext_s0_device, vector_adsorbate_y_rot_cal_Vext_s0_device, vector_adsorbate_z_rot_cal_Vext_s0_device,
                adsorbate_cart_x_rot_cal_Vext_s0_device, adsorbate_cart_y_rot_cal_Vext_s0_device, adsorbate_cart_z_rot_cal_Vext_s0_device, 
                modify_frame_a_cal_Vext_s0_device, modify_frame_b_cal_Vext_s0_device, modify_frame_c_cal_Vext_s0_device,
                minimum_distance_cal_Vext_s0_device,
                V_s0_temp);

    h_offset = (int *) malloc(sizeof(int)*(N_string[0]+1));
    h_offset[0] = 0;
    for (i=1; i<=N_string[0]; i++)
    {
        h_offset[i] = i*N_atom_adsorbate[0]*N_atom_frame[0]*times;
    }
    hipMalloc((void**)&d_offset, (N_string[0]+1)*sizeof(int));
    hipMemcpy(d_offset, h_offset, (N_string[0]+1)*sizeof(int), hipMemcpyHostToDevice);
    free(h_offset);

    d_temp_storage = NULL;
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s0, 
        N_string[0], d_offset, d_offset+1);
    hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s0, 
        N_string[0], d_offset, d_offset+1);
    hipFree(d_temp_storage);
    // hipFree(d_offset);

    // check_double<<<1,32>>>(N_string[0], V_s0);
    // check_double_custom4<<<1,32>>>(401, s0_a_device, s0_b_device, s0_c_device, s0_alpha_device, s0_beta_device, s0_gamma_device, V_s0);
    // hipDeviceSynchronize();
    // return 0;


    // double *temp = (double *) malloc(sizeof(double)*N_string[0]);
    // hipMemcpy(temp, V_s0, (N_string[0])*sizeof(double), hipMemcpyDeviceToHost);
    // for (i=0; i<N_string[0]; i++)
    // {
    //     printf("%lf\n", temp[i]);
    // }

    // hipFree(index_s0_cal_Vext_s0_device);
    // hipFree(index_adsorbate_cal_Vext_s0_device);
    // hipFree(index_frame_cal_Vext_s0_device);
    // hipFree(a_cal_Vext_s0_device);
    // hipFree(b_cal_Vext_s0_device);
    // hipFree(c_cal_Vext_s0_device);
    // hipFree(alpha_rad_cal_Vext_s0_device);
    // hipFree(beta_rad_cal_Vext_s0_device);
    // hipFree(gamma_rad_cal_Vext_s0_device);
    // hipFree(loc_x_cal_Vext_s0_device);
    // hipFree(loc_y_cal_Vext_s0_device);
    // hipFree(loc_z_cal_Vext_s0_device);
    // hipFree(vector_adsorbate_x_rot_cal_Vext_s0_device);
    // hipFree(vector_adsorbate_y_rot_cal_Vext_s0_device);
    // hipFree(vector_adsorbate_z_rot_cal_Vext_s0_device);
    // hipFree(adsorbate_cart_x_rot_cal_Vext_s0_device);
    // hipFree(adsorbate_cart_y_rot_cal_Vext_s0_device);
    // hipFree(adsorbate_cart_z_rot_cal_Vext_s0_device);
    // hipFree(modify_frame_a_cal_Vext_s0_device);
    // hipFree(modify_frame_b_cal_Vext_s0_device);
    // hipFree(modify_frame_c_cal_Vext_s0_device);
    // hipFree(minimum_distance_cal_Vext_s0_device);
    // hipFree(V_s0_temp);
    // hipFree(V_s0);
    double s0_cart_x[1], s0_cart_y[1], s0_cart_z[1];
    double *s0_a_ini, *s0_b_ini, *s0_c_ini;
    double *s0_alpha_ini, *s0_beta_ini, *s0_gamma_ini;
    double *s0_a_final, *s0_b_final, *s0_c_final;
    double *s0_alpha_final, *s0_beta_final, *s0_gamma_final;
    double *s0_x, *s0_y, *s0_z;
    s0_a_ini = (double *) malloc(sizeof(double)*N_string[0]);
    s0_b_ini = (double *) malloc(sizeof(double)*N_string[0]);
    s0_c_ini = (double *) malloc(sizeof(double)*N_string[0]);
    s0_alpha_ini = (double *) malloc(sizeof(double)*N_string[0]);
    s0_beta_ini = (double *) malloc(sizeof(double)*N_string[0]);
    s0_gamma_ini = (double *) malloc(sizeof(double)*N_string[0]);
    s0_a_final = (double *) malloc(sizeof(double)*N_string[0]);
    s0_b_final = (double *) malloc(sizeof(double)*N_string[0]);
    s0_c_final = (double *) malloc(sizeof(double)*N_string[0]);
    s0_alpha_final = (double *) malloc(sizeof(double)*N_string[0]);
    s0_beta_final = (double *) malloc(sizeof(double)*N_string[0]);
    s0_gamma_final = (double *) malloc(sizeof(double)*N_string[0]);
    s0_x = (double *) malloc(sizeof(double)*N_string[0]);
    s0_y = (double *) malloc(sizeof(double)*N_string[0]);
    s0_z = (double *) malloc(sizeof(double)*N_string[0]);

    hipMemcpy(s0_a_ini, s0_a_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_b_ini, s0_b_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_c_ini, s0_c_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_alpha_ini, s0_alpha_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_beta_ini, s0_beta_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_gamma_ini, s0_gamma_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);

    double kb = 1.38e-23, T = 300;
    double *V_s0_1, *V_s0_2;
    double *V_s0_treated, *s0;
    double D_1, D_2;
    V_s0_1 = (double *) malloc(sizeof(double)*N_string[0]);
    V_s0_2 = (double *) malloc(sizeof(double)*N_string[0]);
    V_s0_treated = (double *) malloc(sizeof(double)*N_string[0]);
    s0 = (double *) malloc(sizeof(double)*N_string[0]);
    hipMemcpy(V_s0_1, V_s0, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for (i=0; i<N_string[0]; i++)
    {
        frac2car(s0_a_ini[i], s0_b_ini[i], s0_c_ini[i], frac2car_a, frac2car_b, frac2car_c, s0_cart_x, s0_cart_y, s0_cart_z);
        s0_x[i] = s0_cart_x[0]*1e-10;
        s0_y[i] = s0_cart_y[0]*1e-10;
        s0_z[i] = s0_cart_z[0]*1e-10;
        // printf("%.5e %.5e %.5e\n", s0_a_ini[i], s0_b_ini[i], s0_c_ini[i]);
        // printf("%.5e %.5e %.5e\n", s0_x[i], s0_y[i], s0_z[i]);
        // printf("%lf %lf %lf %lf %lf %lf\n", s0_a_ini[i], s0_b_ini[i], s0_c_ini[i], s0_alpha_ini[i], s0_beta_ini[i], s0_gamma_ini[i]);
    }
    for (i=0; i<N_string[0]; i++)
    {
        if (i==0)
        {
            s0[i] = 0;
        }
        else
        {
            s0[i] = s0[i-1] + sqrt( pow((s0_x[i]-s0_x[i-1]), 2) + pow((s0_y[i]-s0_y[i-1]), 2) + pow((s0_z[i]-s0_z[i-1]), 2) );
        }
         // = 1.0*i/(N_string[0]-1)*1e-10;
        if ((V_s0_1[i]/T)>6e2)
        {
            V_s0_treated[i] = exp(-6e2);
        }
        else
        {
            V_s0_treated[i] = exp(-V_s0_1[i]/T);
        }
        // printf("%.5e\n", V_s0_1[i]);
        
    }
    // printf("length: %.5e\n", s0[N_string[0]-1]);
    switch (direction[0])
    {
        case 1:
            D_1 = 0.5 * pow((La*1e-10), 2) * sqrt((kb*T)/(2*PI*(total_mass_adsorbate/1e3/6.02214076e23))) *  
            (  exp( -max(V_s0_1, N_string[0])/T ) / trapz(s0, V_s0_treated, N_string[0]) );
            break;
        case 2:
            D_1 = 0.5 * pow((Lb*1e-10), 2) * sqrt((kb*T)/(2*PI*(total_mass_adsorbate/1e3/6.02214076e23))) *  
            (  exp( -max(V_s0_1, N_string[0])/T ) / trapz(s0, V_s0_treated, N_string[0]) );
            break;
        case 3:
            D_1 = 0.5 * pow((Lc*1e-10), 2) * sqrt((kb*T)/(2*PI*(total_mass_adsorbate/1e3/6.02214076e23))) *  
            (  exp( -max(V_s0_1, N_string[0])/T ) / trapz(s0, V_s0_treated, N_string[0]) );
            break;
    }





    // remap the string to the set that can calculate the partial derivative
    double *s0_deri_a_device, *s0_deri_b_device, *s0_deri_c_device;
    double *s0_deri_alpha_device, *s0_deri_beta_device, *s0_deri_gamma_device;
    int *s0_deri_index_string_device, *s0_deri_index_var_device;
    int *s0_deri_index_adsorbate_device, *s0_deri_index_frame_device;

    double *s0_deri_loc_x_device, *s0_deri_loc_y_device, *s0_deri_loc_z_device;
    double *s0_deri_vector_adsorbate_x_rot_device, *s0_deri_vector_adsorbate_y_rot_device, *s0_deri_vector_adsorbate_z_rot_device;
    double *s0_deri_adsorbate_cart_x_rot_device, *s0_deri_adsorbate_cart_y_rot_device, *s0_deri_adsorbate_cart_z_rot_device;
    double *s0_deri_modify_frame_a_device, *s0_deri_modify_frame_b_device, *s0_deri_modify_frame_c_device;
    double *s0_deri_minimum_distance_device;
    double *s0_deri_total_Vext_device, *s0_deri_Vext_device;

    double *s0_gradient_device, *s0_gradient_square_device, *s0_gradient_length_device;

    



    double *diff_s_coordinate_all_device, *diff_s_orientation_all_device;
    double *diff_s_coordinate_device, *diff_s_orientation_device;
    double *total_diff_s_coordinate_device, *total_diff_s_orientation_device;


    // double

    // double *a_s0_cal_device, *b_s0_cal_device, *c_s0_cal_device;
    // double *alpha_rad_s0_device, *beta_rad_s0_device, *gamma_rad_s0_device;




    hipMalloc((void **)&s0_deri_a_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_b_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_c_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_alpha_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_beta_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_gamma_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_index_string_device, sizeof(int)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_index_var_device, sizeof(int)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_index_adsorbate_device, sizeof(int)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_index_frame_device, sizeof(int)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_loc_x_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_loc_y_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_loc_z_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_vector_adsorbate_x_rot_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_vector_adsorbate_y_rot_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_vector_adsorbate_z_rot_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_adsorbate_cart_x_rot_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_adsorbate_cart_y_rot_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_adsorbate_cart_z_rot_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_modify_frame_a_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_modify_frame_b_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_modify_frame_c_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_minimum_distance_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_total_Vext_device, sizeof(double)*N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7);
    hipMalloc((void **)&s0_deri_Vext_device, sizeof(double)*N_string[0]*7);
    hipMalloc((void **)&s0_gradient_device, sizeof(double)*N_string[0]*6);
    hipMalloc((void **)&s0_gradient_square_device, sizeof(double)*N_string[0]*6);
    hipMalloc((void **)&s0_gradient_length_device, sizeof(double)*N_string[0]*2);



    hipMalloc((void **)&diff_s_coordinate_all_device, sizeof(double)*N_string[0]*3);
    hipMalloc((void **)&diff_s_orientation_all_device, sizeof(double)*N_string[0]*3);

    hipMalloc((void **)&diff_s_coordinate_device, sizeof(double)*N_string[0]);
    hipMalloc((void **)&diff_s_orientation_device, sizeof(double)*N_string[0]);
    hipMalloc((void **)&total_diff_s_coordinate_device, sizeof(double));
    hipMalloc((void **)&total_diff_s_orientation_device, sizeof(double));



    // parameter used for string method
    double rounding_coeff[1];
    double smooth_coeff[1];
    double *move_angle_rad_device, *move_frac_device;
    double *rounding_coeff_device;
    double *smooth_coeff_device;
    hipMalloc((void **)&move_angle_rad_device, sizeof(double));
    hipMalloc((void **)&move_frac_device, sizeof(double));
    hipMalloc((void **)&rounding_coeff_device, sizeof(double));
    hipMalloc((void **)&smooth_coeff_device, sizeof(double));
    // move_angle_degree[0] = 1.0;
    // move_frac[0] = 1e-4;
    move_angle_rad[0] = 1.0*move_angle_degree[0]/180*PI;
    rounding_coeff[0] = 1e-15;
    smooth_coeff[0] = 1e-4;

    hipMemcpy(move_angle_rad_device, move_angle_rad, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(move_frac_device, move_frac, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rounding_coeff_device, rounding_coeff, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(smooth_coeff_device, smooth_coeff, sizeof(double), hipMemcpyHostToDevice);
    // check_double<<<1,32>>>(1, move_angle_rad_device);
    // check_double<<<1,32>>>(1, move_frac_device);
    // check_double_sci<<<1,32>>>(1, rounding_coeff_device);
    // check_double_sci<<<1,32>>>(1, smooth_coeff_device);
    // hipDeviceSynchronize();
    // return 0;



    int *V_deri_offset = (int *) malloc(sizeof(int)*(N_string[0]*7+1));
    V_deri_offset[0] = 0;
    for (i=1; i<=(N_string[0]*7); i++)
    {
        V_deri_offset[i] = i*N_atom_adsorbate[0]*N_atom_frame[0]*times;
    }
    int *V_deri_offset_device;
    hipMalloc((void**)&V_deri_offset_device, (N_string[0]*7+1)*sizeof(int));
    hipMemcpy(V_deri_offset_device, V_deri_offset, (N_string[0]*7+1)*sizeof(int), hipMemcpyHostToDevice);
    free(V_deri_offset);

    int *s0_gradient_offset = (int *) malloc(sizeof(int)*(N_string[0]*2+1));
    s0_gradient_offset[0] = 0;
    for (i=1; i<=(N_string[0]*2); i++)
    {
        s0_gradient_offset[i] = i*3;
    }
    int *s0_gradient_offset_device;
    hipMalloc((void**)&s0_gradient_offset_device, sizeof(int)*(N_string[0]*2+1));
    hipMemcpy(s0_gradient_offset_device, s0_gradient_offset, sizeof(int)*(N_string[0]*2+1), hipMemcpyHostToDevice);
    free(s0_gradient_offset);


    int *s1_l_sum_1_offset = (int *) malloc(sizeof(int)*(N_string[0]+1));
    s1_l_sum_1_offset[0] = 0;
    for (i=1; i<=(N_string[0]); i++)
    {
        s1_l_sum_1_offset[i] = i*6;
    }
    int *s1_l_sum_1_offset_device;
    hipMalloc((void**)&s1_l_sum_1_offset_device, sizeof(int)*(N_string[0]+1));
    hipMemcpy(s1_l_sum_1_offset_device, s1_l_sum_1_offset, sizeof(int)*(N_string[0]+1), hipMemcpyHostToDevice);
    free(s1_l_sum_1_offset);

    int *s1_l_sum_2_offset = (int *) malloc(sizeof(int)*2);
    s1_l_sum_2_offset[0] = 0;
    s1_l_sum_2_offset[1] = N_string[0];
    int *s1_l_sum_2_offset_device;
    hipMalloc((void**)&s1_l_sum_2_offset_device, sizeof(int)*2);
    hipMemcpy(s1_l_sum_2_offset_device, s1_l_sum_2_offset, sizeof(int)*2, hipMemcpyHostToDevice);
    free(s1_l_sum_2_offset);

    // int *s1_l_cumulation_offset = (int *) malloc(sizeof(int)*(N_string[0]+1));
    // s1_l_cumulation_offset[0] = 0;
    // for (i=1; i<=(N_string[0]); i++)
    // {
    //     s1_l_cumulation_offset[i] = ((int) (i*(i+1)*0.5));
    // }
    // int *s1_l_cumulation_offset_device;
    // hipMalloc((void**)&s1_l_cumulation_offset_device, sizeof(int)*(N_string[0]+1));
    // hipMemcpy(s1_l_cumulation_offset_device, s1_l_cumulation_offset, sizeof(int)*(N_string[0]+1), hipMemcpyHostToDevice);
    // free(s1_l_cumulation_offset);








    int *s1_l_sum_separate_offset = (int *) malloc(sizeof(int)*(N_string[0]+1));
    s1_l_sum_separate_offset[0] = 0;
    for (i=1; i<=(N_string[0]); i++)
    {
        s1_l_sum_separate_offset[i] = i*3;
    }
    int *s1_l_sum_separate_offset_device;
    hipMalloc((void**)&s1_l_sum_separate_offset_device, sizeof(int)*(N_string[0]+1));
    hipMemcpy(s1_l_sum_separate_offset_device, s1_l_sum_separate_offset, sizeof(int)*(N_string[0]+1), hipMemcpyHostToDevice);
    free(s1_l_sum_separate_offset);



    int *s1_l_sum_total_offset = (int *) malloc(sizeof(int)*2);
    s1_l_sum_total_offset[0] = 0;
    s1_l_sum_total_offset[1] = N_string[0];
    int *s1_l_sum_total_offset_device;
    hipMalloc((void**)&s1_l_sum_total_offset_device, sizeof(int)*2);
    hipMemcpy(s1_l_sum_total_offset_device, s1_l_sum_total_offset, sizeof(int)*2, hipMemcpyHostToDevice);
    free(s1_l_sum_total_offset);



    int *s1_l_cumulation_offset = (int *) malloc(sizeof(int)*(N_string[0]+1));
    s1_l_cumulation_offset[0] = 0;
    for (i=1; i<=(N_string[0]); i++)
    {
        s1_l_cumulation_offset[i] = ((int) (i*(i+1)*0.5));
    }
    int *s1_l_cumulation_offset_device;
    hipMalloc((void**)&s1_l_cumulation_offset_device, sizeof(int)*(N_string[0]+1));
    hipMemcpy(s1_l_cumulation_offset_device, s1_l_cumulation_offset, sizeof(int)*(N_string[0]+1), hipMemcpyHostToDevice);
    free(s1_l_cumulation_offset);






    double convergence_coorindate[1];
    double convergence_orientation[1];
    int signal_coordinate[1];
    int signal_orientation[1];
    // printf("check: %lf %lf\n", 1.0*set_conv_trans_percent/100, 1.0*set_conv_rot_percent/100);
    convergence_coorindate[0] = 1.0*set_conv_trans_percent/100*move_frac[0]*N_string[0]*sqrt(3);
    convergence_orientation[0] = 1.0*set_conv_rot_percent/100*move_angle_rad[0]*N_string[0]*sqrt(3);
    signal_coordinate[0] = 0;
    signal_orientation[0] = 0;

    double *convergence_coorindate_device;
    double *convergence_orientation_device;
    int *signal_coordinate_device;
    int *signal_orientation_device;
    hipMalloc((void **)&convergence_coorindate_device, sizeof(double));
    hipMalloc((void **)&convergence_orientation_device, sizeof(double));
    hipMalloc((void **)&signal_coordinate_device, sizeof(int));
    hipMalloc((void **)&signal_orientation_device, sizeof(int));
    hipMemcpy(convergence_coorindate_device, convergence_coorindate, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(convergence_orientation_device, convergence_orientation, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(signal_coordinate_device, signal_coordinate, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(signal_orientation_device, signal_orientation, sizeof(int), hipMemcpyHostToDevice);
    // check_double<<<1,32>>>(1, convergence_coorindate_device);
    // check_double<<<1,32>>>(1, convergence_orientation_device);
    // check_int<<<1,32>>>(1, signal_coordinate_device);
    // check_int<<<1,32>>>(1, signal_orientation_device);
    // hipDeviceSynchronize();
    // return 0;










    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    // all the following items would be included in the iteration/loop
    hipDeviceSynchronize();
    // printf("start\n");
    t = clock();

    int time_set = set_running_step;
    int i_time;
    for (i_time=0; i_time<time_set; i_time++)
    {
        remap_string_var<<<(int)((N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7-1)/running_block_size+1),running_block_size>>>
        (N_atom_adsorbate_device, temp_add_frame_device,

                    N_string_device,

                    s0_a_device, s0_b_device, s0_c_device,
                    s0_alpha_device, s0_beta_device, s0_gamma_device,



                    s0_deri_a_device, s0_deri_b_device, s0_deri_c_device, 
                    s0_deri_alpha_device, s0_deri_beta_device, s0_deri_gamma_device,

                    s0_deri_index_string_device, s0_deri_index_var_device,
                    s0_deri_index_adsorbate_device, s0_deri_index_frame_device,


                    move_angle_rad_device, move_frac_device);







        Vext_s0_deri_cal<<<(int)((N_string[0]*N_atom_adsorbate[0]*N_atom_frame[0]*times*7-1)/running_block_size+1),running_block_size>>>
        (N_atom_adsorbate_device, epsilon_adsorbate_device, sigma_adsorbate_device, 
                    charge_adsorbate_device,
        vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device,
                    N_atom_frame_device, epsilon_frame_device, sigma_frame_device,  
                    charge_frame_device, 
                    frac_a_frame_device, frac_b_frame_device, frac_c_frame_device,
                    times_x_device, times_y_device, times_z_device,
                    cart_x_extended_device, cart_y_extended_device, cart_z_extended_device,
                    frac2car_a_device, frac2car_b_device, frac2car_c_device, 
                    cutoff_device, damping_a_device, 
                    temp_add_frame_device,

                    N_string_device,

                    s0_deri_a_device, s0_deri_b_device, s0_deri_c_device,
                    s0_deri_alpha_device, s0_deri_beta_device, s0_deri_gamma_device,

                    s0_deri_index_adsorbate_device, s0_deri_index_frame_device,

                    s0_deri_loc_x_device, s0_deri_loc_y_device, s0_deri_loc_z_device,
                    s0_deri_vector_adsorbate_x_rot_device, s0_deri_vector_adsorbate_y_rot_device, s0_deri_vector_adsorbate_z_rot_device,
                    s0_deri_adsorbate_cart_x_rot_device, s0_deri_adsorbate_cart_y_rot_device, s0_deri_adsorbate_cart_z_rot_device,
                    s0_deri_modify_frame_a_device, s0_deri_modify_frame_b_device, s0_deri_modify_frame_c_device,
                    s0_deri_minimum_distance_device,

                    s0_deri_total_Vext_device);



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s0_deri_total_Vext_device, s0_deri_Vext_device, 
            N_string[0]*7, V_deri_offset_device, V_deri_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s0_deri_total_Vext_device, s0_deri_Vext_device, 
            N_string[0]*7, V_deri_offset_device, V_deri_offset_device+1);
        hipFree(d_temp_storage);







        s0_grad_cal<<<(int)((N_string[0]*6-1)/running_block_size+1),running_block_size>>>
        (move_frac_device, move_angle_rad_device, rounding_coeff_device,
        N_string_device, s0_deri_Vext_device, s0_gradient_device, s0_gradient_square_device);

        // double *temp;
        // temp = (double *) malloc(sizeof(double)*N_string[0]*6);
        // hipMemcpy(temp, s0_gradient_device, (N_string[0]*6)*sizeof(double), hipMemcpyDeviceToHost);
        // for (i=0; i<N_string[0]; i++)
        // {
        //     printf("%lf %lf %lf %lf %lf %lf\n", temp[i*6+0], temp[i*6+1], temp[i*6+2], temp[i*6+3], temp[i*6+4], temp[i*6+5]);
        // }
        // return 0;


        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s0_gradient_square_device, s0_gradient_length_device, 
            N_string[0]*2, s0_gradient_offset_device, s0_gradient_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s0_gradient_square_device, s0_gradient_length_device, 
            N_string[0]*2, s0_gradient_offset_device, s0_gradient_offset_device+1);
        hipFree(d_temp_storage);



        s0_grad_length_sqrt_cal<<<(int)((N_string[0]*2-1)/running_block_size+1),running_block_size>>>
        (rounding_coeff_device, N_string_device, s0_gradient_length_device);

        // double *temp;
        // temp = (double *) malloc(sizeof(double)*N_string[0]*2);
        // hipMemcpy(temp, s0_gradient_length_device, (N_string[0]*2)*sizeof(double), hipMemcpyDeviceToHost);
        // for (i=0; i<N_string[0]; i++)
        // {
        //     printf("%lf %lf\n", temp[i*2+0], temp[i*2+1]);
        // }
        // return 0;





        s0_new_cal<<<(int)((N_string[0]*6-1)/running_block_size+1),running_block_size>>>
        (rounding_coeff_device, N_string_device, 
        move_frac_device, move_angle_rad_device,




        s0_gradient_length_device, s0_gradient_device,
        s0_a_device, s0_b_device, s0_c_device, 
        s0_alpha_device, s0_beta_device, s0_gamma_device,
        s1_a_device, s1_b_device, s1_c_device,
        s1_alpha_device, s1_beta_device, s1_gamma_device);








        // check_double_custom2<<<1,32>>>(301, s1_a_device, s1_b_device, s1_c_device, s1_alpha_device, s1_beta_device, s1_gamma_device);
        // hipDeviceSynchronize();
        // return 0;


        s1_fix_modify_upgrade<<<(int)((N_string[0]-1)/running_block_size+1),running_block_size>>>
        (N_string_device, 

        direction_device,

        s0_gradient_length_device, s0_gradient_device,
        s0_a_device, s0_b_device, s0_c_device, 
        s0_alpha_device, s0_beta_device, s0_gamma_device,
        s1_a_device, s1_b_device, s1_c_device,
        s1_alpha_device, s1_beta_device, s1_gamma_device);


        s1_frac2cart<<<(int)((N_string[0]-1)/running_block_size+1),running_block_size>>>
        (N_string_device, 

        frac2car_a_device, frac2car_b_device, frac2car_c_device,

        s1_a_device, s1_b_device, s1_c_device,
        s1_cart_x_device, s1_cart_y_device, s1_cart_z_device);



        // check_double_custom2<<<1,32>>>(301, s1_cart_x_device, s1_cart_y_device, s1_cart_z_device, s1_alpha_device, s1_beta_device, s1_gamma_device);
        // check_double_custom2<<<1,32>>>(401, s1_a_device, s1_b_device, s1_c_device, s1_alpha_device, s1_beta_device, s1_gamma_device);
        // hipDeviceSynchronize();
        // return 0;


        
        s1_length_prep<<<(int)((N_string[0]*6-1)/running_block_size+1),running_block_size>>>
        (N_string_device, 

        s1_cart_x_device, s1_cart_y_device, s1_cart_z_device,
        s1_alpha_device, s1_beta_device, s1_gamma_device,
        s1_length_coordinate_all_device, s1_length_orientation_all_device);



        // double *temp1, *temp2;
        // temp1 = (double *) malloc(sizeof(double)*N_string[0]*3);
        // temp2 = (double *) malloc(sizeof(double)*N_string[0]*3);
        // hipMemcpy(temp1, s1_length_coordinate_all_device, (N_string[0]*3)*sizeof(double), hipMemcpyDeviceToHost);
        // hipMemcpy(temp2, s1_length_orientation_all_device, (N_string[0]*3)*sizeof(double), hipMemcpyDeviceToHost);
        // for (i=0; i<N_string[0]; i++)
        // {
        //     printf("%.3e %.3e %.3e %.3e %.3e %.3e\n", temp1[i*3+0], temp1[i*3+1], temp1[i*3+2], temp2[i*3+0], temp2[i*3+1], temp2[i*3+2]);
        // }
        // return 0;



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_coordinate_all_device, s1_length_coordinate_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_coordinate_all_device, s1_length_coordinate_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipFree(d_temp_storage);



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_orientation_all_device, s1_length_orientation_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_orientation_all_device, s1_length_orientation_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipFree(d_temp_storage);



        // check_double_custom<<<1,32>>>(301, s1_length_coordinate_device, s1_length_orientation_device);
        // hipDeviceSynchronize();
        // return 0;



        s1_length_sqrt_cal<<<(int)((N_string[0]*2-1)/running_block_size+1),running_block_size>>>
        (rounding_coeff_device, N_string_device, s1_length_coordinate_device, s1_length_orientation_device);



        // check_double_custom<<<1,32>>>(301, s1_length_coordinate_device, s1_length_orientation_device);
        // hipDeviceSynchronize();
        // return 0;


        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_coordinate_device, s1_l_abs_coordinate_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_coordinate_device, s1_l_abs_coordinate_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipFree(d_temp_storage);



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_orientation_device, s1_l_abs_orientation_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_orientation_device, s1_l_abs_orientation_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipFree(d_temp_storage);



        // check_double_custom<<<1,32>>>(1, s1_l_abs_coordinate_device, s1_l_abs_orientation_device);
        // hipDeviceSynchronize();
        // return 0;



        remap_s1_length_for_cumulation<<<(int)(((N_string[0]*(1+N_string[0])*0.5*2)-1)/running_block_size+1),running_block_size>>>
        (N_string_device, s1_length_coordinate_device, s1_length_orientation_device, 

        s1_length_coordinate_remap_device, s1_length_orientation_remap_device);



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_coordinate_remap_device, s1_length_coordinate_cumulation_device, 
            N_string[0], s1_l_cumulation_offset_device, s1_l_cumulation_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_coordinate_remap_device, s1_length_coordinate_cumulation_device, 
            N_string[0], s1_l_cumulation_offset_device, s1_l_cumulation_offset_device+1);
        hipFree(d_temp_storage);



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_orientation_remap_device, s1_length_orientation_cumulation_device, 
            N_string[0], s1_l_cumulation_offset_device, s1_l_cumulation_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, s1_length_orientation_remap_device, s1_length_orientation_cumulation_device, 
            N_string[0], s1_l_cumulation_offset_device, s1_l_cumulation_offset_device+1);
        hipFree(d_temp_storage);



        // check_double_custom<<<1,32>>>(301, s1_length_coordinate_cumulation_device, s1_length_orientation_cumulation_device);
        // hipDeviceSynchronize();
        // return 0;











        s1_2_s2<<<(int)((N_string[0]*2-1)/running_block_size+1),running_block_size>>>
        (N_string_device, s1_l_abs_coordinate_device, s1_l_abs_orientation_device,
        s1_length_coordinate_cumulation_device, s1_length_orientation_cumulation_device, 

        s1_a_device, s1_b_device, s1_c_device, 
        s1_alpha_device, s1_beta_device, s1_gamma_device,

        s2_a_device, s2_b_device, s2_c_device,
        s2_alpha_device, s2_beta_device, s2_gamma_device);

        // check_double_temp<<<1,32>>>(401, s2_a_device, s2_b_device, s2_c_device);
        // check_double_temp<<<1,32>>>(401, s2_alpha_device, s2_beta_device, s2_gamma_device);
        // check_double_custom2<<<1,32>>>(401, s1_a_device, s1_b_device, s1_c_device, s1_alpha_device, s1_beta_device, s1_gamma_device);
        // hipDeviceSynchronize();
        // return 0;

        // check_double_custom2<<<1,32>>>(401, s0_a_device, s0_b_device, s0_c_device, s0_alpha_device, s0_beta_device, s0_gamma_device);
        // hipDeviceSynchronize();
        // return 0;

        // calculate the potential of initial string
        Vext_cal_s0<<<(int)((N_string[0]*times*N_atom_adsorbate[0]*N_atom_frame[0]-1)/running_block_size+1),running_block_size>>>
        (N_atom_adsorbate_device, epsilon_adsorbate_device, sigma_adsorbate_device, 
                    charge_adsorbate_device, 
        vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device,
                    N_atom_frame_device, epsilon_frame_device, sigma_frame_device, 
                    charge_frame_device, 
                    frac_a_frame_device, frac_b_frame_device, frac_c_frame_device, 
                    times_x_device, times_y_device, times_z_device, 
                    cart_x_extended_device, cart_y_extended_device, cart_z_extended_device, 
                    frac2car_a_device, frac2car_b_device, frac2car_c_device, 
                    cutoff_device, damping_a_device, 
                    temp_add_frame_device, 


                    // int *N_grid_device, int *N_angle_alpha_device, int *N_angle_beta_device, int *N_angle_gamma_device,
                    // double *delta_grid_device, double *delta_angle_device,
                    N_string_device,

                    s0_a_device, s0_b_device, s0_c_device, 
                    s0_alpha_device, s0_beta_device, s0_gamma_device,


                    index_s0_cal_Vext_s0_device,
                    // int *index_a_cal_Vext_s0_device, int *index_b_cal_Vext_s0_device, int *index_c_cal_Vext_s0_device,
                    // int *index_alpha_cal_Vext_s0_device, int *index_beta_cal_Vext_s0_device, int *index_gamma_cal_Vext_s0_device,
                    index_adsorbate_cal_Vext_s0_device, index_frame_cal_Vext_s0_device,

                    a_cal_Vext_s0_device, b_cal_Vext_s0_device, c_cal_Vext_s0_device,
                    alpha_rad_cal_Vext_s0_device, beta_rad_cal_Vext_s0_device, gamma_rad_cal_Vext_s0_device,
                    loc_x_cal_Vext_s0_device, loc_y_cal_Vext_s0_device, loc_z_cal_Vext_s0_device,
                    vector_adsorbate_x_rot_cal_Vext_s0_device, vector_adsorbate_y_rot_cal_Vext_s0_device, vector_adsorbate_z_rot_cal_Vext_s0_device,
                    adsorbate_cart_x_rot_cal_Vext_s0_device, adsorbate_cart_y_rot_cal_Vext_s0_device, adsorbate_cart_z_rot_cal_Vext_s0_device, 
                    modify_frame_a_cal_Vext_s0_device, modify_frame_b_cal_Vext_s0_device, modify_frame_c_cal_Vext_s0_device,
                    minimum_distance_cal_Vext_s0_device,
                    V_s0_temp);
        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s0, 
            N_string[0], d_offset, d_offset+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s0, 
            N_string[0], d_offset, d_offset+1);
        hipFree(d_temp_storage);

        // calculate the potential of moved string
        Vext_cal_s0<<<(int)((N_string[0]*times*N_atom_adsorbate[0]*N_atom_frame[0]-1)/running_block_size+1),running_block_size>>>
        (N_atom_adsorbate_device, epsilon_adsorbate_device, sigma_adsorbate_device, 
                    charge_adsorbate_device, 
        vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device,
                    N_atom_frame_device, epsilon_frame_device, sigma_frame_device, 
                    charge_frame_device, 
                    frac_a_frame_device, frac_b_frame_device, frac_c_frame_device, 
                    times_x_device, times_y_device, times_z_device, 
                    cart_x_extended_device, cart_y_extended_device, cart_z_extended_device, 
                    frac2car_a_device, frac2car_b_device, frac2car_c_device, 
                    cutoff_device, damping_a_device, 
                    temp_add_frame_device, 


                    // int *N_grid_device, int *N_angle_alpha_device, int *N_angle_beta_device, int *N_angle_gamma_device,
                    // double *delta_grid_device, double *delta_angle_device,
                    N_string_device,

                    s2_a_device, s2_b_device, s2_c_device, 
                    s2_alpha_device, s2_beta_device, s2_gamma_device,


                    index_s0_cal_Vext_s0_device,
                    // int *index_a_cal_Vext_s0_device, int *index_b_cal_Vext_s0_device, int *index_c_cal_Vext_s0_device,
                    // int *index_alpha_cal_Vext_s0_device, int *index_beta_cal_Vext_s0_device, int *index_gamma_cal_Vext_s0_device,
                    index_adsorbate_cal_Vext_s0_device, index_frame_cal_Vext_s0_device,

                    a_cal_Vext_s0_device, b_cal_Vext_s0_device, c_cal_Vext_s0_device,
                    alpha_rad_cal_Vext_s0_device, beta_rad_cal_Vext_s0_device, gamma_rad_cal_Vext_s0_device,
                    loc_x_cal_Vext_s0_device, loc_y_cal_Vext_s0_device, loc_z_cal_Vext_s0_device,
                    vector_adsorbate_x_rot_cal_Vext_s0_device, vector_adsorbate_y_rot_cal_Vext_s0_device, vector_adsorbate_z_rot_cal_Vext_s0_device,
                    adsorbate_cart_x_rot_cal_Vext_s0_device, adsorbate_cart_y_rot_cal_Vext_s0_device, adsorbate_cart_z_rot_cal_Vext_s0_device, 
                    modify_frame_a_cal_Vext_s0_device, modify_frame_b_cal_Vext_s0_device, modify_frame_c_cal_Vext_s0_device,
                    minimum_distance_cal_Vext_s0_device,
                    V_s0_temp);
        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s2, 
            N_string[0], d_offset, d_offset+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s2, 
            N_string[0], d_offset, d_offset+1);
        hipFree(d_temp_storage);



        

        check_s2<<<(int)((N_string[0]*3-1)/running_block_size+1),running_block_size>>>
        (N_string_device, V_s0, V_s2,
        s0_alpha_device, s0_beta_device, s0_gamma_device, 
        s2_alpha_device, s2_beta_device, s2_gamma_device);










        smooth_angle<<<(int)((N_string[0]*3-1)/running_block_size+1),running_block_size>>>
        (N_string_device, smooth_coeff_device, 
        s2_alpha_device, s2_beta_device, s2_gamma_device, 
        s2_alpha_smooth_device, s2_beta_smooth_device, s2_gamma_smooth_device);

        hipDeviceSynchronize();
        // return 0;









        // calculate the difference in coordinate and orientation after one iteration
        diff_s_prep<<<(int)((N_string[0]*6-1)/running_block_size+1),running_block_size>>>
        (N_string_device, 

        s0_a_device, s0_b_device, s0_c_device,
        s0_alpha_device, s0_beta_device, s0_gamma_device,

        s2_a_device, s2_b_device, s2_c_device,
        s2_alpha_device, s2_beta_device, s2_gamma_device,

        diff_s_coordinate_all_device, diff_s_orientation_all_device);



        
        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_coordinate_all_device, diff_s_coordinate_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_coordinate_all_device, diff_s_coordinate_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipFree(d_temp_storage);



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_orientation_all_device, diff_s_orientation_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_orientation_all_device, diff_s_orientation_device, 
            N_string[0], s1_l_sum_separate_offset_device, s1_l_sum_separate_offset_device+1);
        hipFree(d_temp_storage);

        hipDeviceSynchronize();
        // check_double_custom<<<1,32>>>
        // (401, diff_s_coordinate_device, diff_s_orientation_device);


        s1_length_sqrt_cal<<<(int)((N_string[0]*2-1)/running_block_size+1),running_block_size>>>
        (rounding_coeff_device, N_string_device, diff_s_coordinate_device, diff_s_orientation_device);


        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_coordinate_device, total_diff_s_coordinate_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_coordinate_device, total_diff_s_coordinate_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipFree(d_temp_storage);



        d_temp_storage = NULL;
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_orientation_device, total_diff_s_orientation_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, diff_s_orientation_device, total_diff_s_orientation_device, 
            1, s1_l_sum_total_offset_device, s1_l_sum_total_offset_device+1);
        hipFree(d_temp_storage);

        if ((i_time%200)==0)
        {   
            check_signal<<<(int)((2-1)/running_block_size+1),running_block_size>>>
            (N_string_device, 
            total_diff_s_coordinate_device, total_diff_s_orientation_device,
            convergence_coorindate_device, convergence_orientation_device,
            signal_coordinate_device, signal_orientation_device);

            hipMemcpy(signal_coordinate, signal_coordinate_device, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(signal_orientation, signal_orientation_device, sizeof(int), hipMemcpyDeviceToHost);

            if ((signal_coordinate[0]==1)&&(signal_orientation[0]==1))
            {
                break;
            }
        }





        copy2s0<<<(int)((N_string[0]*6-1)/running_block_size+1),running_block_size>>>
        (N_string_device, 
        signal_coordinate_device, signal_orientation_device,

        s2_a_device, s2_b_device, s2_c_device, 
        s2_alpha_smooth_device, s2_beta_smooth_device, s2_gamma_smooth_device, 
        s0_a_device, s0_b_device, s0_c_device, 
        s0_alpha_device, s0_beta_device, s0_gamma_device);

        hipDeviceSynchronize();
        
    }

    if ((signal_coordinate[0]==1)&&(signal_orientation[0]==1))
    {
        // printf("converged!\n");
        printf("info:\t1\t%d\t%d\t", i_time, N_atom_frame[0]*times);
    }
    else
    {
        // printf("timed out\n");
        printf("info:\t0\t%d\t%d\t", i_time, N_atom_frame[0]*times);
    }

    // return 0;


    Vext_cal_s0<<<(int)((N_string[0]*times*N_atom_adsorbate[0]*N_atom_frame[0]-1)/running_block_size+1),running_block_size>>>
    (N_atom_adsorbate_device, epsilon_adsorbate_device, sigma_adsorbate_device, 
                charge_adsorbate_device, 
    vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device,
                N_atom_frame_device, epsilon_frame_device, sigma_frame_device, 
                charge_frame_device, 
                frac_a_frame_device, frac_b_frame_device, frac_c_frame_device, 
                times_x_device, times_y_device, times_z_device, 
                cart_x_extended_device, cart_y_extended_device, cart_z_extended_device, 
                frac2car_a_device, frac2car_b_device, frac2car_c_device, 
                cutoff_device, damping_a_device, 
                temp_add_frame_device, 


                // int *N_grid_device, int *N_angle_alpha_device, int *N_angle_beta_device, int *N_angle_gamma_device,
                // double *delta_grid_device, double *delta_angle_device,
                N_string_device,

                s0_a_device, s0_b_device, s0_c_device, 
                s0_alpha_device, s0_beta_device, s0_gamma_device,


                index_s0_cal_Vext_s0_device,
                // int *index_a_cal_Vext_s0_device, int *index_b_cal_Vext_s0_device, int *index_c_cal_Vext_s0_device,
                // int *index_alpha_cal_Vext_s0_device, int *index_beta_cal_Vext_s0_device, int *index_gamma_cal_Vext_s0_device,
                index_adsorbate_cal_Vext_s0_device, index_frame_cal_Vext_s0_device,

                a_cal_Vext_s0_device, b_cal_Vext_s0_device, c_cal_Vext_s0_device,
                alpha_rad_cal_Vext_s0_device, beta_rad_cal_Vext_s0_device, gamma_rad_cal_Vext_s0_device,
                loc_x_cal_Vext_s0_device, loc_y_cal_Vext_s0_device, loc_z_cal_Vext_s0_device,
                vector_adsorbate_x_rot_cal_Vext_s0_device, vector_adsorbate_y_rot_cal_Vext_s0_device, vector_adsorbate_z_rot_cal_Vext_s0_device,
                adsorbate_cart_x_rot_cal_Vext_s0_device, adsorbate_cart_y_rot_cal_Vext_s0_device, adsorbate_cart_z_rot_cal_Vext_s0_device, 
                modify_frame_a_cal_Vext_s0_device, modify_frame_b_cal_Vext_s0_device, modify_frame_c_cal_Vext_s0_device,
                minimum_distance_cal_Vext_s0_device,
                V_s0_temp);

    // h_offset = (int *) malloc(sizeof(int)*(N_string[0]+1));
    // h_offset[0] = 0;
    // for (i=1; i<=N_string[0]; i++)
    // {
    //     h_offset[i] = i*N_atom_adsorbate[0]*N_atom_frame[0]*times;
    // }
    // hipMalloc((void**)&d_offset, (N_string[0]+1)*sizeof(int));
    // hipMemcpy(d_offset, h_offset, (N_string[0]+1)*sizeof(int), hipMemcpyHostToDevice);
    // free(h_offset);

    d_temp_storage = NULL;
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s0, 
        N_string[0], d_offset, d_offset+1);
    hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, V_s0_temp, V_s0, 
        N_string[0], d_offset, d_offset+1);
    hipFree(d_temp_storage);
    // hipFree(d_offset);







    hipDeviceSynchronize();
    // // print xyz
    // check_double_special<<<1,32>>>
    // (401, N_atom_adsorbate_device,

    // frac2car_a_device, frac2car_b_device, frac2car_c_device,

    // vector_adsorbate_x_device, vector_adsorbate_y_device, vector_adsorbate_z_device,

    // s0_a_device, s0_b_device, s0_c_device, 
    // s0_alpha_device, s0_beta_device, s0_gamma_device);
    // hipDeviceSynchronize();

    t = clock() - t;



    hipMemcpy(s0_a_final, s0_a_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_b_final, s0_b_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_c_final, s0_c_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_alpha_final, s0_alpha_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_beta_final, s0_beta_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(s0_gamma_final, s0_gamma_device, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipMemcpy(V_s0_2, V_s0, sizeof(double)*N_string[0], hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    for (i=0; i<N_string[0]; i++)
    {
        frac2car(s0_a_final[i], s0_b_final[i], s0_c_final[i], frac2car_a, frac2car_b, frac2car_c, s0_cart_x, s0_cart_y, s0_cart_z);
        s0_x[i] = s0_cart_x[0]*1e-10;
        s0_y[i] = s0_cart_y[0]*1e-10;
        s0_z[i] = s0_cart_z[0]*1e-10;
    }
    for (i=0; i<N_string[0]; i++)
    {
        if (i==0)
        {
            s0[i] = 0;
        }
        else
        {
            s0[i] = s0[i-1] + sqrt( pow((s0_x[i]-s0_x[i-1]), 2) + pow((s0_y[i]-s0_y[i-1]), 2) + pow((s0_z[i]-s0_z[i-1]), 2) );
        }

        if ((V_s0_2[i]/T)>6e2)
        {
            V_s0_treated[i] = exp(-6e2);
        }
        else
        {
            V_s0_treated[i] = exp(-V_s0_2[i]/T);
        }   
    }
    // printf("length: %.5e\n", s0[N_string[0]-1]);
    switch (direction[0])
    {
        case 1:
            D_2 = 0.5 * pow((La*1e-10), 2) * sqrt((kb*T)/(2*PI*(total_mass_adsorbate/1e3/6.02214076e23))) *  
            (  exp( -max(V_s0_2, N_string[0])/T ) / trapz(s0, V_s0_treated, N_string[0]) );
            break;
        case 2:
            D_2 = 0.5 * pow((Lb*1e-10), 2) * sqrt((kb*T)/(2*PI*(total_mass_adsorbate/1e3/6.02214076e23))) *  
            (  exp( -max(V_s0_2, N_string[0])/T ) / trapz(s0, V_s0_treated, N_string[0]) );
            break;
        case 3:
            D_2 = 0.5 * pow((Lc*1e-10), 2) * sqrt((kb*T)/(2*PI*(total_mass_adsorbate/1e3/6.02214076e23))) *  
            (  exp( -max(V_s0_2, N_string[0])/T ) / trapz(s0, V_s0_treated, N_string[0]) );
            break;
    }



    if (argc==3)
    {
        fp1 =fopen(argv[2], "w+");
        if (D_1>D_2)
        {
            for (i=0; i<N_string[0]; i++)
            {
                fprintf(fp1, "%lf %lf %lf %lf %lf %lf %lf\n", s0_a_ini[i], s0_b_ini[i], s0_c_ini[i], s0_alpha_ini[i], s0_beta_ini[i], s0_gamma_ini[i], V_s0_1[i]);
            }
        }
        else
        {
            for (i=0; i<N_string[0]; i++)
            {
                fprintf(fp1, "%lf %lf %lf %lf %lf %lf %lf\n", s0_a_final[i], s0_b_final[i], s0_c_final[i], s0_alpha_final[i], s0_beta_final[i], s0_gamma_final[i], V_s0_2[i]);
            }
        }
        
        fclose(fp1);
    }
    else if (argc==4)
    {
        fp1 =fopen(argv[3], "w+");
        if (D_1>D_2)
        {
            for (i=0; i<N_string[0]; i++)
            {
                fprintf(fp1, "%lf %lf %lf %lf %lf %lf %lf\n", s0_a_ini[i], s0_b_ini[i], s0_c_ini[i], s0_alpha_ini[i], s0_beta_ini[i], s0_gamma_ini[i], V_s0_1[i]);
            }
        }
        else
        {
            for (i=0; i<N_string[0]; i++)
            {
                fprintf(fp1, "%lf %lf %lf %lf %lf %lf %lf\n", s0_a_final[i], s0_b_final[i], s0_c_final[i], s0_alpha_final[i], s0_beta_final[i], s0_gamma_final[i], V_s0_2[i]);
            }
        }
        fclose(fp1);
    }

    printf("%lf\n", ((double)t)/CLOCKS_PER_SEC);

}
